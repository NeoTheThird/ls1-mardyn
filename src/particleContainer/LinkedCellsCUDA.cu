#include "hip/hip_runtime.h"
// Andreas Kirsch 2010

#include "LinkedCellsCUDA.h"
#include "molecules/Molecule.h"
#include "cutil_math.h"
#include "math.h"

double3 operator +=( double3 &a, const double3 &b ) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
}

double3 operator -=( double3 &a, const double3 &b ) {
	a.x -= b.x;
	a.y -= b.y;
	a.z -= b.z;
}

double3 operator -( const double3 &a, const double3 &b ) {
	return make_double3( a.x - b.x, a.y - b.y, a.z - b.z );
}

double3 operator *( const double &s, const double3 &b ) {
	return make_double3( s * b.x, s * b.y, s * b.z );
}

#define OUT

#define CUDA_TIMING

#ifdef CUDA_TIMING
class CUDATimer {
private:
	hipEvent_t _startEvent, _endEvent;

public:
	CUDATimer() {
		CUDA_THROW_ON_ERROR( hipEventCreate( &_startEvent ) );
		CUDA_THROW_ON_ERROR( hipEventCreate( &_endEvent ) );
	}

	~CUDATimer() {
		CUDA_THROW_ON_ERROR( hipEventDestroy( _startEvent ) );
		CUDA_THROW_ON_ERROR( hipEventDestroy( _endEvent ) );
	}

	void begin() {
		CUDA_THROW_ON_ERROR( hipEventRecord( _startEvent ) );
	}

	void end() {
		CUDA_THROW_ON_ERROR( hipEventRecord( _endEvent ) );
	}

	float getElapsedTime() {
		CUDA_THROW_ON_ERROR( hipEventSynchronize( _endEvent ) );

		float elapsedTime;
		CUDA_THROW_ON_ERROR( hipEventElapsedTime( &elapsedTime, _startEvent, _endEvent ) );

		return elapsedTime;
	}

	void printElapsedTime( const char *format ) {
		printf( format, getElapsedTime() );
	}
};
#else
class CUDATimer {
public:
	void begin() {
	}

	void end() {
	}

	float getElapsedTime() {
		return 0.0f;
	}

	void printElapsedTime( const char *format ) {
	}
};
#endif

__device__ void calculateLennardJones( const CUDAPrecisionType3 distance, const CUDAPrecisionType distanceSquared, CUDAPrecisionType epsilon, CUDAPrecisionType sigmaSquared,
		OUT CUDAPrecisionType3 &force, OUT CUDAPrecisionType &potential) {
	CUDAPrecisionType invdr2 = 1.f / distanceSquared;
	CUDAPrecisionType lj6 = sigmaSquared * invdr2; lj6 = lj6 * lj6 * lj6;
	CUDAPrecisionType lj12 = lj6 * lj6;
	CUDAPrecisionType lj12m6 = lj12 - lj6;
	potential = 4.0f * epsilon * lj12m6;
	// result: force = fac * distance = fac * |distance| * normalized(distance)
	CUDAPrecisionType fac = -24.0f * epsilon * (lj12 + lj12m6) * invdr2;
	force = fac * distance;
}

__device__ int getCellIndex( int startIndex, int2 dimension, int3 gridOffsets ) {
	const int idx = blockIdx.x;
	const int3 gridIndex = make_int3(
			idx % dimension.x,
			(idx / dimension.x) % dimension.y,
			idx / dimension.x / dimension.y
		);
	const int cellIndex = startIndex + dot( gridIndex, gridOffsets );

	return cellIndex;
}

#define MEASURE_ERROR
#define USE_REF

//#define TEST_CELL_COVERAGE
#ifdef TEST_CELL_COVERAGE
#	include "LinkedCellsCUDAcellCoverage.cum"
#else
#	ifndef USE_REF
#		include "LinkedCellsCUDAfast.cum"
#	else
#		include "LinkedCellsCUDAref.cum"
#endif
#endif

void LinkedCellsCUDA::traversePairs() {
#ifdef MEASURE_ERROR
	_linkedCells.traversePairs();

	CUDAPrecisionType cpuPotential = _domain.getLocalUpot();
	CUDAPrecisionType cpuVirial = _domain.getLocalVirial();
	printf( "CPU Potential: %f CPU Virial: %f\n", cpuPotential, cpuVirial );
#endif
	LinkedCellsCUDA_Internal::DomainValues domainValues;
	_cudaInternal.calculateForces( domainValues );
	// update the domain values
	_domain.setLocalUpot( domainValues.potential );
	_domain.setLocalVirial( domainValues.virial );
}

void LinkedCellsCUDA_Internal::calculateForces( LinkedCellsCUDA_Internal::DomainValues &domainValues ) {
	manageAllocations();

	initComponentInfos();
	initCellInfosAndCopyPositions();
	prepareDeviceMemory();

	calculateAllLJFoces();

	extractResultsFromDeviceMemory();
	reducePotentialAndVirial( domainValues.potential, domainValues.virial );

	printf( "Potential: %f Virial: %f\n", domainValues.potential, domainValues.virial );
	printf( "Average Potential: %f Average Virial: %f\n", domainValues.potential / _numLJCenters, domainValues.virial / _numLJCenters );

#ifdef MEASURE_ERROR
	determineForceError();
#endif

	//updateMoleculeForces();
}

void LinkedCellsCUDA_Internal::manageAllocations()
{
	// HACK HACK HACK
	_numLJCenters = 2 * _linkedCells.getParticles().size();
	_numCells = _linkedCells.getCells().size();

	// TODO: use memalign like the old code?
	if( _numLJCenters > _maxLJCenters ) {
		_positions.resize( _numLJCenters );
		_forces.resize( _numLJCenters );
		_componentLJCenterIndices.resize( _numLJCenters );

		_maxLJCenters = _numLJCenters;
	}

	if( _numCells > _maxCells ) {
		_cellStartIndices.resize( _numCells + 1 );
		_domainValues.resize( _numCells );

		_maxCells = _numCells;
	}
}

void LinkedCellsCUDA_Internal::freeAllocations()
{
	_positions.resize( 0 );
	_forces.resize( 0 );
	_componentLJCenterIndices.resize( 0 );

	_cellStartIndices.resize( 0 );
	_domainValues.resize( 0 );

	_componentLJCenterInfos.resize( 0 );
	_componentLJCenterOffsetFromFirst.resize( 0 );
	delete[] _componentStartIndices;
}

void LinkedCellsCUDA_Internal::initComponentInfos() {
	const std::vector< Component > &components = _domain.getComponents();
	_componentStartIndices = new int[ components.size() ];

	// TODO: clean up this code..
	// initialize _numComponentLJCenters and _componentStartIndices
	_numComponentLJCenters = 0;
	for( int i = 0 ; i < components.size() ; i++ ) {
		_componentStartIndices[ i ] = _numComponentLJCenters;
		_numComponentLJCenters += components[i].numLJcenters();
	}

	// initialize _componentLJCenterOffsetFromFirst
	_componentLJCenterOffsetFromFirst.resize( _numComponentLJCenters);
	for( int i = 0 ; i < components.size() ; i++ ) {
		for( int j = 0 ; j < components[i].numLJcenters() ; j++ ) {
			_componentLJCenterOffsetFromFirst[ _componentStartIndices[i] + j ] = j;
		}
	}

	// initialize _componentLJCenterInfos
	_componentLJCenterInfos.resize( _numComponentLJCenters * _numComponentLJCenters );

	for( int indexCompA = 0 ; indexCompA < components.size() ; indexCompA++ ) {
		const Component &compA = components[ indexCompA ];
		assert( compA.numLJcenters() <= 2 );

		for( int indexCompB = 0 ; indexCompB < components.size() ; indexCompB++ ) {
			const Component &compB = components[ indexCompB ];
			assert( compB.numLJcenters() <= 2 );

			for( int indexLJCenterA = 0 ; indexLJCenterA < compA.numLJcenters() ; indexLJCenterA++ ) {
				const LJcenter &ljCenterA = compA.ljcenter(indexLJCenterA);

				for( int indexLJCenterB = 0 ; indexLJCenterB < compB.numLJcenters() ; indexLJCenterB++ ) {
					const LJcenter &ljCenterB = compB.ljcenter(indexLJCenterB);

					const int targetIndex = (_componentStartIndices[indexCompA] + indexLJCenterA) * _numComponentLJCenters +
							_componentStartIndices[indexCompB] + indexLJCenterB;
					ComponentLJCenterInfo &ljCenterInfo = _componentLJCenterInfos[ targetIndex ];

					ljCenterInfo.epsilon = sqrt( ljCenterA.eps() * ljCenterB.eps() );
					CUDAPrecisionType sigma = 0.5f * ( ljCenterA.sigma() + ljCenterB.sigma() );
					ljCenterInfo.sigmaSquared = sigma * sigma;
				}
			}
		}
	}

	_componentLJCenterOffsetFromFirst.copyToDevice();
	_componentLJCenterInfos.copyToDevice();
}

void LinkedCellsCUDA_Internal::initCellInfosAndCopyPositions()
{
	int currentIndex = 0;
	for( int i = 0 ; i < _numCells ; i++ ) {
		const Cell &cell = _linkedCells.getCells()[i];

		_cellStartIndices[i] = currentIndex;

		const std::list<Molecule*> &particles = cell.getParticlePointers();
		for( std::list<Molecule*>::const_iterator iterator = particles.begin() ; iterator != particles.end() ; iterator++ ) {
			Molecule &molecule = **iterator;

			const unsigned int numLJCenters = molecule.numLJcenters();
			if( numLJCenters > 2 ) {
				printf( "%i has more than 2 lj centers!\n", currentIndex );
			}

			const Quaternion &q = molecule.q();

			for( int ljCenterIndex = 0 ; ljCenterIndex < numLJCenters ; ljCenterIndex++ ) {
				const double *ljCenterInitialPosition = _domain.getComponents()[ molecule.componentid() ].ljcenter( ljCenterIndex ).r();
				double ljCenterRelativePosition[3];
				q.rotateinv( ljCenterInitialPosition, ljCenterRelativePosition );

				_positions[currentIndex].x = molecule.r(0) + ljCenterRelativePosition[0];
				_positions[currentIndex].y = molecule.r(1) + ljCenterRelativePosition[1];
				_positions[currentIndex].z = molecule.r(2) + ljCenterRelativePosition[2];

				_componentLJCenterIndices[currentIndex] = _componentStartIndices[ molecule.componentid() ] + ljCenterIndex;
				currentIndex++;
			}
		}
	}

	_cellStartIndices[_numCells] = currentIndex;
}

void LinkedCellsCUDA_Internal::prepareDeviceMemory()
{
	// TODO: use page-locked/mapped memory
	int3 *dimensions = (int3*) _linkedCells.getCellDimensions();
	printf( "Num LJ Centers: %i Num Cells: %i (%i x %i x %i)\n", _numLJCenters, _numCells, dimensions->x, dimensions->y, dimensions->z );

	CUDATimer copyTimer;

	copyTimer.begin();

	// copy the input data to the device
	_positions.copyToDevice();
	_componentLJCenterIndices.copyToDevice();
	_cellStartIndices.copyToDevice();

	// reset the output buffers
	_forces.zeroDevice();
	_domainValues.zeroDevice();

	copyTimer.end();
	copyTimer.printElapsedTime( "host to device copying: %f ms\n" );
}

void LinkedCellsCUDA_Internal::extractResultsFromDeviceMemory() {
	_forces.copyToHost();
	_domainValues.copyToHost();
}

void LinkedCellsCUDA_Internal::updateMoleculeForces() {
	int currentIndex = 0;
	for( int i = 0 ; i < _numCells ; i++ ) {
		const Cell &cell = _linkedCells.getCells()[i];

		const std::list<Molecule*> &particles = cell.getParticlePointers();
		for( std::list<Molecule*>::const_iterator iterator = particles.begin() ; iterator != particles.end() ; iterator++ ) {
			Molecule &molecule = **iterator;

			for( int i = 0 ; i < molecule.numLJcenters() ; i++ ) {
				molecule.Fljcenterset( i, (CUDAPrecisionType*) &_forces[currentIndex++] );
			}
		}
	}
}

static double lengthd( const CUDAPrecisionType3 &v ) {
	double lengthSquared = (double)v.x*v.x + v.y*v.y + v.z*v.z;
	return sqrt(lengthSquared);
}

void LinkedCellsCUDA_Internal::determineForceError() {
	double totalError = 0.0;
	double totalRelativeError = 0.0;
	float epsilon = 5.96e-06f;

	double avgCPUMagnitude = 0.0, avgCUDAMagnitude = 0.0;
	int currentIndex = 0;
	for( int i = 0 ; i < _numCells ; i++ ) {
		const Cell &cell = _linkedCells.getCells()[i];

		const std::list<Molecule*> &particles = cell.getParticlePointers();
		for( std::list<Molecule*>::const_iterator iterator = particles.begin() ; iterator != particles.end() ; iterator++ ) {
		    Molecule &molecule = **iterator;
			for( int ljCenter = 0 ; ljCenter < molecule.numLJcenters() ; ljCenter++ ) {
				CUDAPrecisionType3 &cudaForce = _forces[currentIndex++];

				if( !cell.isBoundaryCell() && !cell.isInnerCell() ) {
					continue;
				}

				const double *cpuForceD = molecule.ljcenter_F(ljCenter);
				CUDAPrecisionType3 cpuForce = make_float3( cpuForceD[0], cpuForceD[1], cpuForceD[2] );
				CUDAPrecisionType3 deltaForce = cudaForce - cpuForce;

				CUDAPrecisionType cpuForceLength = lengthd( cpuForce );
				CUDAPrecisionType cudaForceLength = lengthd( cudaForce );
				CUDAPrecisionType deltaForceLength = lengthd( deltaForce );

				if( isfinite(cpuForceLength) && isfinite( cudaForceLength ) && isfinite( deltaForceLength ) ) {
					avgCPUMagnitude += cpuForceLength;
					avgCUDAMagnitude += cudaForceLength;

					totalError += deltaForceLength;

					if( cpuForceLength > epsilon ) {
						double relativeError = deltaForceLength / cpuForceLength;
						totalRelativeError += relativeError;
					}
				}
				else {
					;
				}
			}
		}
	}

	avgCPUMagnitude /= currentIndex;
	avgCUDAMagnitude /= currentIndex;

	printf( "Average CPU Mag:  %f\n"
			"Average CUDA Mag: %f\n"
			"Average Error: %f\n"
			"Average Relative Error: %f\n", avgCPUMagnitude, avgCUDAMagnitude, totalError / _numLJCenters, totalRelativeError / _numLJCenters );
}

void LinkedCellsCUDA_Internal::calculateAllLJFoces() {
	CUDATimer singleCellsTimer, cellPairsTimer;

	// TODO: wtf? this is from the old code
	const float cutOffRadiusSquared = _cutOffRadius * _cutOffRadius;

	const dim3 blockSize = dim3( WARP_SIZE, NUM_WARPS, 1 );

	// intra cell forces
	const int *dimensions = _linkedCells.getCellDimensions();
	assert( dimensions[0] >= 2 && dimensions[1] >= 2 && dimensions[2] >=2 );

	const int3 zero3 = {0,0,0};
	const int3 xDirection = {1,0,0};
	const int3 yDirection = {0,1,0};
	const int3 zDirection = {0,0,1};
	// always make sure that each direction contains one component == 1
	const int3 directions[] = {
			{1,0,0},{0,1,0},{0,0,1},
			{1,1,0},{1,0,1},{0,1,1},
			{-1,1,0},{-1,0,1},{0,-1,1},
			{-1,1,1},{1,-1,1},{1,1,-1},
			{1,1,1}
	};

	cellPairsTimer.begin();

	for( int i = 0 ; i < sizeof( directions ) / sizeof( directions[0] ) ; i++ ) {
		const int3 &direction = directions[i];
		// we are going to iterate over odd and even slices (either xy-, xz- or yz-slices)

		// define: the main direction is the normal of the slice plane

		int neighborOffset = getDirectionOffset( direction );

		// contains the oriented direction as if the main direction was (0,0,1)
		int3 localDirection;
		// dimensions as if the main direction was (0,0,1)
		int3 localDimensions;
		int3 gridOffsets;

		// determine the direction of the plane (xy, xz or yz)
		if( direction.x == 1 ) {
			// yz plane (main direction: x)
			localDirection = make_int3( direction.y, direction.z, direction.x );
			localDimensions = make_int3( dimensions[1], dimensions[2], dimensions[0] );
			gridOffsets = make_int3(
					getDirectionOffset( yDirection ),
					getDirectionOffset( zDirection ),
					getDirectionOffset( xDirection )
				);
		}
		else if( direction.y == 1 ) {
			// xz plane (main direction: y)
			localDirection = make_int3( direction.x, direction.z, direction.y );
			localDimensions = make_int3( dimensions[0], dimensions[2], dimensions[1] );
			gridOffsets = make_int3(
					getDirectionOffset( xDirection ),
					getDirectionOffset( zDirection ),
					getDirectionOffset( yDirection )
				);
		}
		else if( direction.z == 1 ) {
			// xy plane (main direction: z)
			localDirection = direction;
			localDimensions = make_int3( dimensions[0], dimensions[1], dimensions[2] );
			gridOffsets = make_int3(
					getDirectionOffset( xDirection ),
					getDirectionOffset( yDirection ),
					getDirectionOffset( zDirection )
				);
		}
		else {
			assert( false );
		}

		// determine the startOffset as first cell near (0,0,0) so that start + neighborOffset won't be out of bounds
		int evenSlicesStartIndex = getCellOffset( -min( direction, zero3 ) );
		// odd slices start one slice "down"
		int oddSlicesStartIndex = evenSlicesStartIndex + gridOffsets.z;

		// set z to 0
		// adapt the local dimensions in such a way as to avoid out of bounds accesses at the "far corners"
		// the positive components of localSliceDirection affect the max corner of the slice
		// the negative ones the min corner (see *StartIndex). dimensions = max - min => use abs to count both correctly.
		localDimensions -= abs( localDirection );

		// always move 2 slices in local z direction, so we hit either odd or even slices in one kernel call
		gridOffsets.z *= 2;

		// there are floor( dimZ / 2 ) odd slices
		int numOddSlices = localDimensions.z / 2;
		int numEvenSlices = localDimensions.z - numOddSlices;

		int numCellsInSlice = localDimensions.x * localDimensions.y;

		// do all even slices
		Kernel_calculatePairLJForces<<<numEvenSlices * numCellsInSlice, blockSize>>>(
				_positions.devicePtr(), _componentLJCenterIndices.devicePtr(), _forces.devicePtr(),
				_componentLJCenterInfos.devicePtr(), _numComponentLJCenters,
				_cellStartIndices.devicePtr(), _domainValues.devicePtr(),
				evenSlicesStartIndex, make_int2( localDimensions ), gridOffsets,
				neighborOffset,
				cutOffRadiusSquared
			);

		// do all odd slices
		Kernel_calculatePairLJForces<<<numOddSlices * numCellsInSlice, blockSize>>>(
				_positions.devicePtr(), _componentLJCenterIndices.devicePtr(), _forces.devicePtr(),
				_componentLJCenterInfos.devicePtr(), _numComponentLJCenters,
				_cellStartIndices.devicePtr(), _domainValues.devicePtr(),
				oddSlicesStartIndex, make_int2( localDimensions ), gridOffsets,
				neighborOffset,
				cutOffRadiusSquared
			);
	}

	cellPairsTimer.end();

	// inner cell forces
	singleCellsTimer.begin();

	Kernel_calculateInnerLJForces<<<_numCells, blockSize>>>(
			_positions.devicePtr(), _componentLJCenterIndices.devicePtr(), _forces.devicePtr(),
			_componentLJCenterInfos.devicePtr(), _componentLJCenterOffsetFromFirst.devicePtr(), _numComponentLJCenters,
			_cellStartIndices.devicePtr(), _domainValues.devicePtr(),
			cutOffRadiusSquared
		);

	singleCellsTimer.end();

	singleCellsTimer.printElapsedTime( "intra cell LJ forces: %f ms " );
	cellPairsTimer.printElapsedTime( "inter cell LJ forces: %f ms\n" );
}

void LinkedCellsCUDA_Internal::reducePotentialAndVirial( OUT CUDAPrecisionType &potential, OUT CUDAPrecisionType &virial ) {
	const std::vector<unsigned long> &innerCellIndices = _linkedCells.getInnerCellIndices();
	const std::vector<unsigned long> &boundaryCellIndices = _linkedCells.getBoundaryCellIndices();

	potential = 0.0f;
	virial = 0.0f;
	for( int i = 0 ; i < innerCellIndices.size() ; i++ ) {
		int innerCellIndex = innerCellIndices[i];
#ifdef TEST_CELL_COVERAGE
		if( (int) _domainValues[ innerCellIndex ].x != 26 ) {
			printf( "%i (badly covered inner cell - coverage: %f)\n", innerCellIndex, _domainValues[ innerCellIndex ].x );
		}
#endif
		potential += _domainValues[ innerCellIndex ].x;
		virial += _domainValues[ innerCellIndex ].y;
	}
	for( int i = 0 ; i < boundaryCellIndices.size() ; i++ ) {
		int boundaryCellIndex = boundaryCellIndices[ i ];

#ifdef TEST_CELL_COVERAGE
		if( (int) _domainValues[ boundaryCellIndex ].x != 26 ) {
			printf( "%i (badly covered inner cell - coverage: %f)\n", boundaryCellIndex, _domainValues[ boundaryCellIndex ].x );
		}
#endif

		potential += _domainValues[ boundaryCellIndex ].x;
		virial += _domainValues[ boundaryCellIndex ].y;
	}

	// every contribution is added twice so divide by 2
	potential /= 2.0f;
	virial /= 2.0f;

	// TODO: I have no idea why the sign is different in the GPU code...
	virial = -virial;
}
