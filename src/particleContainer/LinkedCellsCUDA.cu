#include "hip/hip_runtime.h"
// Andreas Kirsch 2010

#include "LinkedCellsCUDA.h"
#include "molecules/Molecule.h"
#include "cutil_math.h"

#define OUT

#define CUDA_TIMING

#ifdef CUDA_TIMING
class CUDATimer {
private:
	hipEvent_t _startEvent, _endEvent;

public:
	CUDATimer() {
		CUDA_THROW_ON_ERROR( hipEventCreate( &_startEvent ) );
		CUDA_THROW_ON_ERROR( hipEventCreate( &_endEvent ) );
	}

	~CUDATimer() {
		CUDA_THROW_ON_ERROR( hipEventDestroy( _startEvent ) );
		CUDA_THROW_ON_ERROR( hipEventDestroy( _endEvent ) );
	}

	void begin() {
		CUDA_THROW_ON_ERROR( hipEventRecord( _startEvent ) );
	}

	void end() {
		CUDA_THROW_ON_ERROR( hipEventRecord( _endEvent ) );
	}

	float getElapsedTime() {
		CUDA_THROW_ON_ERROR( hipEventSynchronize( _endEvent ) );

		float elapsedTime;
		CUDA_THROW_ON_ERROR( hipEventElapsedTime( &elapsedTime, _startEvent, _endEvent ) );

		return elapsedTime;
	}

	void printElapsedTime( const char *format ) {
		printf( format, getElapsedTime() );
	}
};
#else
class CUDATimer {
public:
	void begin() {
	}

	void end() {
	}

	float getElapsedTime() {
		return 0.0f;
	}

	void printElapsedTime( const char *format ) {
	}
};
#endif

__device__ void calculateLennardJones( const float3 distance, const float distanceSquared, float epsilon, float sigmaSquared,
		OUT float3 &force, OUT float &potential) {
	float invdr2 = 1.f / distanceSquared;
	float lj6 = sigmaSquared * invdr2; lj6 = lj6 * lj6 * lj6;
	float lj12 = lj6 * lj6;
	float lj12m6 = lj12 - lj6;
	potential = 4.0f * epsilon * lj12m6;
	// result: force = fac * distance = fac * |distance| * normalized(distance)
	float fac = -24.0f * epsilon * (lj12 + lj12m6) * invdr2;
	force = fac * distance;
}

__device__ int getCellIndex( int startIndex, int2 dimension, int3 gridOffsets ) {
	const int idx = blockIdx.x;
	const int3 gridIndex = make_int3(
			idx % dimension.x,
			(idx / dimension.x) % dimension.y,
			idx / dimension.x / dimension.y
		);
	const int cellIndex = startIndex + dot( gridIndex, gridOffsets );

	return cellIndex;
}


#define MAX_BLOCK_SIZE 512
#define WARP_SIZE 32
#define NUM_WARPS 8
#define BLOCK_SIZE (WARP_SIZE*NUM_WARPS)

// = ceil( a / b )
__device__ inline int iceil(int a, int b) {
	return (a+b-1) / b;
}

// = b if a % b = 0, a % b otherwise
__device__ inline int shiftedMod( int a, int b ) {
	int r = a % b;
	return (r > 0) ? r : b;
}

__global__ void Kernel_calculateInnerLJForces( float3 *positions, OUT float3 *forces, int2 *cellInfos, OUT float2 *domainValues,
		float epsilon, float sigmaSquared, float cutOffRadiusSquared ) {
	const int cellIndex = blockIdx.x;

	const int cellStart = cellInfos[cellIndex].x;
	const int cellLength = cellInfos[cellIndex].y;

	__shared__ float totalThreadPotential[NUM_WARPS][WARP_SIZE];
	__shared__ float totalThreadVirial[NUM_WARPS][WARP_SIZE];

	totalThreadPotential[threadIdx.y][threadIdx.x] = 0.0f;
	totalThreadVirial[threadIdx.y][threadIdx.x] = 0.0f;

	__shared__ float3 cachedBForces[NUM_WARPS][WARP_SIZE];
	__shared__ float3 cachedBPositions[NUM_WARPS][WARP_SIZE];

	const int transferSize = BLOCK_SIZE;
	const int numTransfers = iceil( cellLength, transferSize );

	for( int transferIndex = 0 ; transferIndex < numTransfers ; transferIndex++ ) {
		const int transferCellIndex = cellStart + transferIndex * transferSize + WARP_SIZE * threadIdx.y + threadIdx.x;
		bool validCell = false;

		// load data into cache
		if( transferCellIndex < cellLength ) {
			validCell = true;
			cachedBForces[threadIdx.y][threadIdx.x] = forces[transferCellIndex];
			cachedBPositions[threadIdx.y][threadIdx.x] = positions[transferCellIndex];
		}

		// I'm working on WARP_SIZE many data entries at once during processing, so there should be a natural synchronization?
		// TODO: remove this __syncthreads maybe?
		__syncthreads();

		if( validCell ) {
			const int aIndex = WARP_SIZE * threadIdx.y + threadIdx.x;
			const float3 &aPosition = ((float3*)cachedBPositions)[aIndex];

			for( int bIndex = 0; bIndex < aIndex ; bIndex++ ) {
				const float3 &bPosition = ((float3*)cachedBPositions)[bIndex];

				const float3 distance = bPosition - aPosition;
				const float distanceSquared = dot( distance, distance );
				if( distanceSquared > cutOffRadiusSquared ) {
					continue;
				}

				float3 force;
				float potential;
				calculateLennardJones( distance, distanceSquared, epsilon, sigmaSquared, force, potential );

				totalThreadPotential[threadIdx.y][threadIdx.x] += potential;
				float virial = dot( force, distance );
				totalThreadVirial[threadIdx.y][threadIdx.x] += virial;

				((float3*)cachedBForces)[aIndex] += force;
				((float3*)cachedBForces)[bIndex] -= force;
			}
		}

		// push data back
		if( transferCellIndex < cellLength ) {
			forces[transferCellIndex] = cachedBForces[threadIdx.y][threadIdx.x];
		}
	}

	__syncthreads();

	for( int transferIndexA = 1 ; transferIndexA < numTransfers ; transferIndexA++ ) {
		const int aTransferCellIndex = cellStart + transferIndexA * transferSize + WARP_SIZE * threadIdx.y + threadIdx.x;

		bool validACell = false;
		float3 cachedAForce;
		float3 cachedAPosition;

		// load A data into (register) cache
		if( aTransferCellIndex < cellLength ) {
			validACell = true;
			cachedAForce = forces[aTransferCellIndex];
			cachedAPosition = positions[aTransferCellIndex];
		}

		for( int transferIndexB = 0 ; transferIndexB < transferIndexA ; transferIndexB++ ) {
			const int bTransferCellIndex = cellStart + transferIndexB * transferSize + WARP_SIZE * threadIdx.y + threadIdx.x;

			// load B data into cache
			if( bTransferCellIndex < cellLength ) {
				cachedBForces[threadIdx.y][threadIdx.x] = forces[bTransferCellIndex];
				cachedBPositions[threadIdx.y][threadIdx.x] = positions[bTransferCellIndex];
			}

			// I'm working on WARP_SIZE many data entries at once during processing, so there should be a natural synchronization?
			// TODO: remove this __syncthreads maybe?
			__syncthreads();

			// process block
			if( validACell ) {
				const int numBWarps = (transferIndexB < numTransfers - 1) ? NUM_WARPS : iceil( shiftedMod( cellLength, transferSize ), WARP_SIZE );
				for( int warpShiftIndex = 0 ; warpShiftIndex < numBWarps ; warpShiftIndex++ ) {
					const int bWarpIndex = (threadIdx.y + warpShiftIndex) % numBWarps;

					const int numBCells = (bWarpIndex < numBWarps - 1) ? WARP_SIZE : shiftedMod(cellLength, WARP_SIZE);
					for( int cellShiftIndex = 0 ; cellShiftIndex < numBCells ; cellShiftIndex++ ) {
						const int bThreadIndex = (threadIdx.x + cellShiftIndex) % numBCells;

						const float3 &bPosition = cachedBPositions[bWarpIndex][bThreadIndex];

						const float3 distance = bPosition - cachedAPosition;
						const float distanceSquared = dot( distance, distance );
						if( distanceSquared > cutOffRadiusSquared ) {
							continue;
						}

						float3 force;
						float potential;
						calculateLennardJones( distance, distanceSquared, epsilon, sigmaSquared, force, potential );

						totalThreadPotential[threadIdx.y][threadIdx.x] += potential;
						float virial = dot( force, distance );
						totalThreadVirial[threadIdx.y][threadIdx.x] += virial;

						cachedAForce += force;
						cachedBForces[bWarpIndex][bThreadIndex] -= force;
					}
				}
			}

			// push B data back
			if( bTransferCellIndex < cellLength ) {
				forces[bTransferCellIndex] = cachedBForces[threadIdx.y][threadIdx.x];
			}
		}

		// push A data back
		if( aTransferCellIndex < cellLength ) {
			forces[aTransferCellIndex] = cachedAForce;
		}
	}

	// reduce the potential and the virial
	// ASSERT: BLOCK_SIZE is power of 2
	for( int power = 2 ; power <= BLOCK_SIZE ; power <<= 1 ) {
		__syncthreads();

		const int index = WARP_SIZE * threadIdx.y + threadIdx.x;
		if( (index & (power-1)) == 0 ) {
			const int neighborIndex = index + (power >> 1);

			((float*)totalThreadPotential)[index] += ((float*)totalThreadPotential)[neighborIndex];
			((float*)totalThreadVirial)[index] += ((float*)totalThreadVirial)[neighborIndex];
		}
	}

	if( threadIdx.x == 0 && threadIdx.y == 0 ) {
		domainValues[cellIndex].x = totalThreadPotential[0][0] * 2;
		domainValues[cellIndex].y = totalThreadVirial[0][0] * 2;
	}
}


// threadIdx.xy = intraWarpIndex | warpIndex
__global__ void Kernel_calculatePairLJForces( float3 *positions, OUT float3 *forces, int2 *cellInfos, OUT float2 *domainValues,
		int startIndex, int2 dimension, int3 gridOffsets,
		int neighborOffset,
		float epsilon, float sigmaSquared, float cutOffRadiusSquared ) {
	int cellIndex = getCellIndex( startIndex, dimension, gridOffsets );
	int neighborIndex = cellIndex + neighborOffset;

	// ensure that cellA_length <= cellB_length (which will use fewer data transfers)
	// (numTransfersA + numTransfersA * numTransfersB) * transferSize
	if( cellInfos[cellIndex].y > cellInfos[neighborIndex].y ) {
		// swap cellIndex and neighborIndex
		cellIndex = neighborIndex;
		neighborIndex -= neighborOffset;
	}

	const int cellA_start = cellInfos[cellIndex].x;
	const int cellA_length = cellInfos[cellIndex].y;
	const int cellB_start = cellInfos[neighborIndex].x;
	const int cellB_length = cellInfos[neighborIndex].y;

	__shared__ float totalThreadPotential[NUM_WARPS][WARP_SIZE];
	__shared__ float totalThreadVirial[NUM_WARPS][WARP_SIZE];

	totalThreadPotential[threadIdx.y][threadIdx.x] = 0.0f;
	totalThreadVirial[threadIdx.y][threadIdx.x] = 0.0f;

	__shared__ float3 cachedBForces[NUM_WARPS][WARP_SIZE];
	__shared__ float3 cachedBPositions[NUM_WARPS][WARP_SIZE];

	const int transferSize = BLOCK_SIZE;
	const int numTransfersA = iceil( cellA_length, transferSize );
	const int numTransfersB = iceil( cellB_length, transferSize );

	for( int transferIndexA = 0 ; transferIndexA < numTransfersA ; transferIndexA++ ) {
		const int numAWarps = (transferIndexA < numTransfersA - 1) ? NUM_WARPS : iceil( shiftedMod( cellA_length, transferSize ), WARP_SIZE );
		const int aTransferCellIndex = cellA_start + transferIndexA * transferSize + WARP_SIZE * threadIdx.y + threadIdx.x;

		bool validACell = false;
		float3 cachedAForce;
		float3 cachedAPosition;

		// load A data into (register) cache
		if( aTransferCellIndex < cellA_length ) {
			validACell = true;
			cachedAForce = forces[aTransferCellIndex];
			cachedAPosition = positions[aTransferCellIndex];
		}

		for( int transferIndexB = 0 ; transferIndexB < numTransfersB ; transferIndexB++ ) {
			const int bTransferCellIndex = cellB_start + transferIndexB * transferSize + WARP_SIZE * threadIdx.y + threadIdx.x;

			// load B data into cache
			if( bTransferCellIndex < cellB_length ) {
				cachedBForces[threadIdx.y][threadIdx.x] = forces[bTransferCellIndex];
				cachedBPositions[threadIdx.y][threadIdx.x] = positions[bTransferCellIndex];
			}

			// I'm working on WARP_SIZE many data entries at once during processing, so there should be a natural synchronization?
			// TODO: remove this __syncthreads maybe?
			__syncthreads();

			// process block
			if( validACell ) {
				const int numBWarps = (transferIndexB < numTransfersB - 1) ? NUM_WARPS : iceil( shiftedMod( cellB_length, transferSize ), WARP_SIZE );
				for( int warpShiftIndex = 0 ; warpShiftIndex < numBWarps ; warpShiftIndex++ ) {
					const int bWarpIndex = (threadIdx.y + warpShiftIndex) % numBWarps;

					const int numBCells = (bWarpIndex < numBWarps - 1) ? WARP_SIZE : shiftedMod(cellB_length, WARP_SIZE);
					for( int cellShiftIndex = 0 ; cellShiftIndex < numBCells ; cellShiftIndex++ ) {
						const int bThreadIndex = (threadIdx.x + cellShiftIndex) % numBCells;

						const float3 &bPosition = cachedBPositions[bWarpIndex][bThreadIndex];

						const float3 distance = bPosition - cachedAPosition;
						const float distanceSquared = dot( distance, distance );
						if( distanceSquared > cutOffRadiusSquared ) {
							continue;
						}

						float3 force;
						float potential;
						calculateLennardJones( distance, distanceSquared, epsilon, sigmaSquared, force, potential );

						totalThreadPotential[threadIdx.y][threadIdx.x] += potential;
						float virial = dot( force, distance );
						totalThreadVirial[threadIdx.y][threadIdx.x] += virial;

						cachedAForce += force;
						cachedBForces[bWarpIndex][bThreadIndex] -= force;
					}
				}
			}

			// push B data back
			if( bTransferCellIndex < cellB_length ) {
				forces[bTransferCellIndex] = cachedBForces[threadIdx.y][threadIdx.x];
			}
		}

		// push A data back
		if( aTransferCellIndex < cellA_length ) {
			forces[aTransferCellIndex] = cachedAForce;
		}
	}

	// reduce the potential and the virial
	// ASSERT: BLOCK_SIZE is power of 2
	for( int power = 2 ; power <= BLOCK_SIZE ; power <<= 1 ) {
		__syncthreads();

		const int index = WARP_SIZE * threadIdx.y + threadIdx.x;
		if( (index & (power-1)) == 0 ) {
			const int neighborIndex = index + (power >> 1);

			((float*)totalThreadPotential)[index] += ((float*)totalThreadPotential)[neighborIndex];
			((float*)totalThreadVirial)[index] += ((float*)totalThreadVirial)[neighborIndex];
		}
	}

	if( threadIdx.x == 0 && threadIdx.y == 0 ) {
		domainValues[cellIndex].x += totalThreadPotential[0][0];
		domainValues[cellIndex].y += totalThreadVirial[0][0];
		domainValues[neighborIndex].x += totalThreadPotential[0][0];
		domainValues[neighborIndex].y += totalThreadVirial[0][0];
	}
}

LinkedCellsCUDA_Internal::DomainValues LinkedCellsCUDA_Internal::calculateForces() {
	manageAllocations();

	initCellInfosAndCopyPositions();
	prepareDeviceMemory();

	calculateAllLJFoces();

	DomainValues domainValues;
	extractResultsFromDeviceMemory();
	reducePotentialAndVirial( domainValues.potential, domainValues.virial );

	printf( "Potential: %f Virial: %f\n", domainValues.potential, domainValues.virial );
	printf( "Average Potential: %f Average Virial: %f\n", domainValues.potential / _numParticles, domainValues.virial / _numParticles );

	determineForceError();

	updateMoleculeForces();

	return domainValues;
}

void LinkedCellsCUDA_Internal::manageAllocations()
{
	_numParticles = _linkedCells.getParticles().size();
	_numCells = _linkedCells.getCells().size();

	// TODO: use memalign like the old code?
	if( _numParticles > _maxParticles ) {
		_positions.resize( _numParticles );
		_forces.resize( _numParticles );

		_maxParticles = _numParticles;
	}

	if( _numCells > _maxCells ) {
		_cellInfos.resize( _numCells );
		_domainValues.resize( _numCells );

		_maxCells = _numCells;
	}
}

void LinkedCellsCUDA_Internal::freeAllocations()
{
	_positions.resize( 0 );
	_forces.resize( 0 );

	_cellInfos.resize( 0 );
	_domainValues.resize( 0 );
}

void LinkedCellsCUDA_Internal::initCellInfosAndCopyPositions()
{
	int currentIndex = 0;
	for( int i = 0 ; i < _numCells ; i++ ) {
		const Cell &cell = _linkedCells.getCells()[i];

		_cellInfos[i].x = currentIndex;
		_cellInfos[i].y = cell.getMoleculeCount();

		const std::list<Molecule*> &particles = cell.getParticlePointers();
		for( std::list<Molecule*>::const_iterator iterator = particles.begin() ; iterator != particles.end() ; iterator++ ) {
			Molecule &molecule = **iterator;
			_positions[currentIndex].x = molecule.r(0);
			_positions[currentIndex].y = molecule.r(1);
			_positions[currentIndex].z = molecule.r(2);
			currentIndex++;
		}
	}
}

void LinkedCellsCUDA_Internal::prepareDeviceMemory()
{
	// TODO: use page-locked/mapped memory
	printf( "%i\n", _numParticles );

	CUDATimer copyTimer;

	copyTimer.begin();

	_positions.copyToDevice();
	_cellInfos.copyToDevice();

	// init device forces to 0
	_forces.zeroDevice();
	// not needed: _domainValues.zeroDevice();

	copyTimer.end();
	copyTimer.printElapsedTime( "host to device copying: %f ms\n" );
}

void LinkedCellsCUDA_Internal::extractResultsFromDeviceMemory() {
	_forces.copyToHost();
	_domainValues.copyToHost();
}

void LinkedCellsCUDA_Internal::updateMoleculeForces() {
	int currentIndex = 0;
	for( int i = 0 ; i < _numCells ; i++ ) {
		const Cell &cell = _linkedCells.getCells()[i];

		const std::list<Molecule*> &particles = cell.getParticlePointers();
		for( std::list<Molecule*>::const_iterator iterator = particles.begin() ; iterator != particles.end() ; iterator++ ) {
			Molecule &molecule = **iterator;
			molecule.Fljcenterset( 0, (float*) &_forces[currentIndex] );
			currentIndex++;
		}
	}
}

void LinkedCellsCUDA_Internal::determineForceError() {
	double totalError = 0.0;
	double totalRelativeError = 0.0;
	float epsilon = 5.96e-06f;

	float avgCPUMagnitude = 0.0, avgCUDAMagnitude = 0.0;
	int currentIndex = 0;
	for( int i = 0 ; i < _numCells ; i++ ) {
		const Cell &cell = _linkedCells.getCells()[i];

		const std::list<Molecule*> &particles = cell.getParticlePointers();
		for( std::list<Molecule*>::const_iterator iterator = particles.begin() ; iterator != particles.end() ; iterator++ ) {
			Molecule &molecule = **iterator;
			float3 &cudaForce = _forces[currentIndex];
			const double *cpuForceD = molecule.ljcenter_F(0);
			float3 cpuForce = make_float3( cpuForceD[0], cpuForceD[1], cpuForceD[2] );
			float3 deltaForce = cudaForce - cpuForce;

			avgCPUMagnitude += length( cpuForce );
			avgCUDAMagnitude += length( cudaForce );

			float error = length( deltaForce );
			totalError += error;

			if( error > epsilon ) {
				float relativeError = error / length( cpuForce );
				totalRelativeError += relativeError;
			}

			currentIndex++;
		}
	}

	avgCPUMagnitude /= _numParticles;
	avgCUDAMagnitude /= _numParticles;

	printf( "Average CPU Mag:  %f\n"
			"Average CUDA Mag: %f\n"
			"Average Error: %f\n"
			"Average Relative Error: %f\n", avgCPUMagnitude, avgCUDAMagnitude, totalError / _numParticles, totalRelativeError / _numParticles );
}

void LinkedCellsCUDA_Internal::calculateAllLJFoces() {
	CUDATimer singleCellsTimer, cellPairsTimer;

	// TODO: wtf? this is from the old code
	const float epsilon = 1.0f;
	const float sigmaSquared = 1.0f;
	const float cutOffRadiusSquared = _cutOffRadius * _cutOffRadius;

	const dim3 blockSize = dim3( WARP_SIZE, NUM_WARPS, 1 );

	singleCellsTimer.begin();

	// inner forces first
	Kernel_calculateInnerLJForces<<<_numCells, blockSize>>>(
			_positions.devicePtr(), _forces.devicePtr(),_cellInfos.devicePtr(), _domainValues.devicePtr(),
			epsilon, sigmaSquared, cutOffRadiusSquared
		);

	singleCellsTimer.end();

	// pair forces
	const int *dimensions = _linkedCells.getCellDimensions();
	assert( dimensions[0] >= 2 && dimensions[1] >= 2 && dimensions[2] >=2 );

	const int3 zero3 = {0,0,0};
	const int3 xDirection = {1,0,0};
	const int3 yDirection = {0,1,0};
	const int3 zDirection = {0,0,1};
	// always make sure that each direction contains one component == 1
	const int3 directions[] = {
			{1,0,0},{0,1,0},{0,0,1},
			{1,1,0},{1,0,1},{0,1,1},
			{-1,1,0},{-1,0,1},{0,-1,1},
			{-1,1,1},{1,-1,1},{1,1,-1},
			{1,1,1}
	};

	cellPairsTimer.begin();

	for( int i = 0 ; i < sizeof( directions ) / sizeof( directions[0] ) ; i++ ) {
		const int3 &direction = directions[i];
		// we are going to iterate over odd and even slices (either xy-, xz- or yz-slices)

		// define: the main direction is the normal of the slice plane

		int neighborOffset = getDirectionOffset( direction );

		// contains the oriented direction as if the main direction was (0,0,1)
		int3 localDirection;
		// dimensions as if the main direction was (0,0,1)
		int3 localDimensions;
		int3 gridOffsets;

		// determine the direction of the plane (xy, xz or yz)
		if( direction.x == 1 ) {
			// yz plane (main direction: x)
			localDirection = make_int3( direction.y, direction.z, direction.x );
			localDimensions = make_int3( dimensions[1], dimensions[2], dimensions[0] );
			gridOffsets = make_int3(
					getDirectionOffset( yDirection ),
					getDirectionOffset( zDirection ),
					getDirectionOffset( xDirection )
				);
		}
		else if( direction.y == 1 ) {
			// xz plane (main direction: y)
			localDirection = make_int3( direction.x, direction.z, direction.y );
			localDimensions = make_int3( dimensions[0], dimensions[2], dimensions[1] );
			gridOffsets = make_int3(
					getDirectionOffset( xDirection ),
					getDirectionOffset( zDirection ),
					getDirectionOffset( yDirection )
				);
		}
		else if( direction.z == 1 ) {
			// xy plane (main direction: z)
			localDirection = direction;
			localDimensions = make_int3( dimensions[0], dimensions[1], dimensions[2] );
			gridOffsets = make_int3(
					getDirectionOffset( xDirection ),
					getDirectionOffset( yDirection ),
					getDirectionOffset( zDirection )
				);
		}
		else {
			assert( false );
		}

		// determine the startOffset as first cell near (0,0,0) so that start + neighborOffset won't be out of bounds
		int evenSlicesStartIndex = getCellOffset( -min( direction, zero3 ) );
		// odd slices start one slice "down"
		int oddSlicesStartIndex = evenSlicesStartIndex + gridOffsets.z;

		// set z to 0
		// adapt the local dimensions in such a way as to avoid out of bounds accesses at the "far corners"
		// the positive components of localSliceDirection affect the max corner of the slice
		// the negative ones the min corner (see *StartIndex). dimensions = max - min => use abs to count both correctly.
		localDimensions -= abs( localDirection );

		// always move 2 slices in local z direction, so we hit either odd or even slices in one kernel call
		gridOffsets.z *= 2;

		// there are floor( dimZ / 2 ) odd slices
		int numOddSlices = localDimensions.z / 2;
		int numEvenSlices = localDimensions.z - numOddSlices;

		int numCellsInSlice = localDimensions.x * localDimensions.y;

		// do all even slices
		Kernel_calculatePairLJForces<<<numEvenSlices * numCellsInSlice, blockSize>>>(
				_positions.devicePtr(), _forces.devicePtr(),_cellInfos.devicePtr(), _domainValues.devicePtr(),
				evenSlicesStartIndex, make_int2( localDimensions ), gridOffsets,
				neighborOffset,
				epsilon, sigmaSquared, cutOffRadiusSquared
			);

		// do all odd slices
		Kernel_calculatePairLJForces<<<numOddSlices * numCellsInSlice, blockSize>>>(
				_positions.devicePtr(), _forces.devicePtr(),_cellInfos.devicePtr(), _domainValues.devicePtr(),
				oddSlicesStartIndex, make_int2( localDimensions ), gridOffsets,
				neighborOffset,
				epsilon, sigmaSquared, cutOffRadiusSquared
			);
	}

	cellPairsTimer.end();

	singleCellsTimer.printElapsedTime( "intra cell LJ forces: %f ms " );
	cellPairsTimer.printElapsedTime( "inter cell LJ forces: %f ms\n" );
}

void LinkedCellsCUDA_Internal::reducePotentialAndVirial( OUT float &potential, OUT float &virial ) {
	const std::vector<unsigned long> &innerCellIndices = _linkedCells.getInnerCellIndices();
	const std::vector<unsigned long> &boundaryCellIndices = _linkedCells.getBoundaryCellIndices();

	potential = 0.0f;
	for( int i = 0 ; i < innerCellIndices.size() ; i++ ) {
		potential += _domainValues[ i ].x;
		virial += _domainValues[ i ].y;
	}
	for( int i = 0 ; i < boundaryCellIndices.size() ; i++ ) {
		potential += _domainValues[ i ].x;
		virial += _domainValues[ i ].y;
	}

	// every contribution is added twice so divide by 2
	potential /= 2.0f;
	virial /= 2.0f;

	// TODO: I have no idea why the sign is different in the GPU code...
	virial = -virial;
}
