#include "hip/hip_runtime.h"
// Andreas Kirsch 2010

#include "LinkedCellsCUDA.h"
#include "molecules/Molecule.h"
#include "cutil_math.h"

#define OUT

#define CUDA_TIMING

#ifdef CUDA_TIMING
class CUDATimer {
private:
	hipEvent_t _startEvent, _endEvent;

public:
	CUDATimer() {
		CUDA_THROW_ON_ERROR( hipEventCreate( &_startEvent ) );
		CUDA_THROW_ON_ERROR( hipEventCreate( &_endEvent ) );
	}

	~CUDATimer() {
		CUDA_THROW_ON_ERROR( hipEventDestroy( _startEvent ) );
		CUDA_THROW_ON_ERROR( hipEventDestroy( _endEvent ) );
	}

	void begin() {
		CUDA_THROW_ON_ERROR( hipEventRecord( _startEvent ) );
	}

	void end() {
		CUDA_THROW_ON_ERROR( hipEventRecord( _endEvent ) );
	}

	float getElapsedTime() {
		CUDA_THROW_ON_ERROR( hipEventSynchronize( _endEvent ) );

		float elapsedTime;
		CUDA_THROW_ON_ERROR( hipEventElapsedTime( &elapsedTime, _startEvent, _endEvent ) );

		return elapsedTime;
	}

	void printElapsedTime( const char *format ) {
		printf( format, getElapsedTime() );
	}
};
#else
class CUDATimer {
public:
	void begin() {
	}

	void end() {
	}

	float getElapsedTime() {
		return 0.0f;
	}

	void printElapsedTime( const char *format ) {
	}
};
#endif

__device__ void calculateLennardJones( const float3 distance, const float distanceSquared, float epsilon, float sigmaSquared,
		OUT float3 &force, OUT float &potential) {
	float invdr2 = 1.f / distanceSquared;
	float lj6 = sigmaSquared * invdr2; lj6 = lj6 * lj6 * lj6;
	float lj12 = lj6 * lj6;
	float lj12m6 = lj12 - lj6;
	potential = 4.0f * epsilon * lj12m6;
	// result: force = fac * distance = fac * |distance| * normalized(distance)
	float fac = -24.0f * epsilon * (lj12 + lj12m6) * invdr2;
	force = fac * distance;
}

__device__ int getCellIndex( int startIndex, int2 dimension, int3 gridOffsets ) {
	const int idx = blockIdx.x;
	const int3 gridIndex = make_int3(
			idx % dimension.x,
			(idx / dimension.x) % dimension.y,
			idx / dimension.x / dimension.y
		);
	const int cellIndex = startIndex + dot( gridIndex, gridOffsets );

	return cellIndex;
}

//#define TEST_CELL_COVERAGE
#ifdef TEST_CELL_COVERAGE
#include "LinkedCellsCUDAcellCoverage.cum"
#else
#include "LinkedCellsCUDAfast.cum"
#endif
//#include "LinkedCellsCUDAref.cum"

LinkedCellsCUDA_Internal::DomainValues LinkedCellsCUDA_Internal::calculateForces() {
	manageAllocations();

	initCellInfosAndCopyPositions();
	prepareDeviceMemory();

	calculateAllLJFoces();

	DomainValues domainValues;
	extractResultsFromDeviceMemory();
	reducePotentialAndVirial( domainValues.potential, domainValues.virial );

	printf( "Potential: %f Virial: %f\n", domainValues.potential, domainValues.virial );
	printf( "Average Potential: %f Average Virial: %f\n", domainValues.potential / _numParticles, domainValues.virial / _numParticles );

	determineForceError();

	updateMoleculeForces();

	return domainValues;
}

void LinkedCellsCUDA_Internal::manageAllocations()
{
	_numParticles = _linkedCells.getParticles().size();
	_numCells = _linkedCells.getCells().size();

	// TODO: use memalign like the old code?
	if( _numParticles > _maxParticles ) {
		_positions.resize( _numParticles );
		_forces.resize( _numParticles );

		_maxParticles = _numParticles;
	}

	if( _numCells > _maxCells ) {
		_cellInfos.resize( _numCells );
		_domainValues.resize( _numCells );

		_maxCells = _numCells;
	}
}

void LinkedCellsCUDA_Internal::freeAllocations()
{
	_positions.resize( 0 );
	_forces.resize( 0 );

	_cellInfos.resize( 0 );
	_domainValues.resize( 0 );
}

void LinkedCellsCUDA_Internal::initCellInfosAndCopyPositions()
{
	int currentIndex = 0;
	for( int i = 0 ; i < _numCells ; i++ ) {
		const Cell &cell = _linkedCells.getCells()[i];

		_cellInfos[i].x = currentIndex;
		_cellInfos[i].y = cell.getMoleculeCount();

		const std::list<Molecule*> &particles = cell.getParticlePointers();
		for( std::list<Molecule*>::const_iterator iterator = particles.begin() ; iterator != particles.end() ; iterator++ ) {
			Molecule &molecule = **iterator;

			const unsigned int numLJCenters = molecule.numLJcenters();
			if( numLJCenters > 1 ) {
				printf( "%i has more than 1 lj center!\n", currentIndex );
			}

			_positions[currentIndex].x = molecule.r(0);
			_positions[currentIndex].y = molecule.r(1);
			_positions[currentIndex].z = molecule.r(2);

			currentIndex++;
		}
	}
}

void LinkedCellsCUDA_Internal::prepareDeviceMemory()
{
	// TODO: use page-locked/mapped memory
	int3 *dimensions = (int3*) _linkedCells.getCellDimensions();
	printf( "Num Particles: %i Num Cells: %i (%i x %i x %i)\n", _numParticles, _numCells, dimensions->x, dimensions->y, dimensions->z );

	CUDATimer copyTimer;

	copyTimer.begin();

	_positions.copyToDevice();
	_cellInfos.copyToDevice();

	// init device forces to 0
	_forces.zeroDevice();
	// not needed: _domainValues.zeroDevice();

	copyTimer.end();
	copyTimer.printElapsedTime( "host to device copying: %f ms\n" );
}

void LinkedCellsCUDA_Internal::extractResultsFromDeviceMemory() {
	_forces.copyToHost();
	_domainValues.copyToHost();
}

void LinkedCellsCUDA_Internal::updateMoleculeForces() {
	int currentIndex = 0;
	for( int i = 0 ; i < _numCells ; i++ ) {
		const Cell &cell = _linkedCells.getCells()[i];

		const std::list<Molecule*> &particles = cell.getParticlePointers();
		for( std::list<Molecule*>::const_iterator iterator = particles.begin() ; iterator != particles.end() ; iterator++ ) {
			Molecule &molecule = **iterator;
			molecule.Fljcenterset( 0, (float*) &_forces[currentIndex] );
			currentIndex++;
		}
	}
}

void LinkedCellsCUDA_Internal::determineForceError() {
	double totalError = 0.0;
	double totalRelativeError = 0.0;
	float epsilon = 5.96e-06f;

	float avgCPUMagnitude = 0.0, avgCUDAMagnitude = 0.0;
	int currentIndex = 0;
	for( int i = 0 ; i < _numCells ; i++ ) {
		const Cell &cell = _linkedCells.getCells()[i];

		const std::list<Molecule*> &particles = cell.getParticlePointers();
		for( std::list<Molecule*>::const_iterator iterator = particles.begin() ; iterator != particles.end() ; iterator++, currentIndex++ ) {
			if( !cell.isBoundaryCell() && !cell.isInnerCell() ) {
				continue;
			}

			Molecule &molecule = **iterator;
			float3 &cudaForce = _forces[currentIndex];
			const double *cpuForceD = molecule.ljcenter_F(0);
			float3 cpuForce = make_float3( cpuForceD[0], cpuForceD[1], cpuForceD[2] );
			float3 deltaForce = cudaForce - cpuForce;

			avgCPUMagnitude += length( cpuForce );
			avgCUDAMagnitude += length( cudaForce );

			float error = length( deltaForce );
			totalError += error;

			if( error > epsilon ) {
				float relativeError = error / length( cpuForce );
				totalRelativeError += relativeError;
			}
		}
	}

	avgCPUMagnitude /= currentIndex;
	avgCUDAMagnitude /= currentIndex;

	printf( "Average CPU Mag:  %f\n"
			"Average CUDA Mag: %f\n"
			"Average Error: %f\n"
			"Average Relative Error: %f\n", avgCPUMagnitude, avgCUDAMagnitude, totalError / _numParticles, totalRelativeError / _numParticles );
}

void LinkedCellsCUDA_Internal::calculateAllLJFoces() {
	CUDATimer singleCellsTimer, cellPairsTimer;

	// TODO: wtf? this is from the old code
	const float epsilon = 1.0f;
	const float sigmaSquared = 1.0f;
	const float cutOffRadiusSquared = _cutOffRadius * _cutOffRadius;

	const dim3 blockSize = dim3( WARP_SIZE, NUM_WARPS, 1 );

	singleCellsTimer.begin();

	// inner forces first
	Kernel_calculateInnerLJForces<<<_numCells, blockSize>>>(
			_positions.devicePtr(), _forces.devicePtr(),_cellInfos.devicePtr(), _domainValues.devicePtr(),
			epsilon, sigmaSquared, cutOffRadiusSquared
		);

	singleCellsTimer.end();

	// pair forces
	const int *dimensions = _linkedCells.getCellDimensions();
	assert( dimensions[0] >= 2 && dimensions[1] >= 2 && dimensions[2] >=2 );

	const int3 zero3 = {0,0,0};
	const int3 xDirection = {1,0,0};
	const int3 yDirection = {0,1,0};
	const int3 zDirection = {0,0,1};
	// always make sure that each direction contains one component == 1
	const int3 directions[] = {
			{1,0,0},{0,1,0},{0,0,1},
			{1,1,0},{1,0,1},{0,1,1},
			{-1,1,0},{-1,0,1},{0,-1,1},
			{-1,1,1},{1,-1,1},{1,1,-1},
			{1,1,1}
	};

	cellPairsTimer.begin();

	for( int i = 0 ; i < sizeof( directions ) / sizeof( directions[0] ) ; i++ ) {
		const int3 &direction = directions[i];
		// we are going to iterate over odd and even slices (either xy-, xz- or yz-slices)

		// define: the main direction is the normal of the slice plane

		int neighborOffset = getDirectionOffset( direction );

		// contains the oriented direction as if the main direction was (0,0,1)
		int3 localDirection;
		// dimensions as if the main direction was (0,0,1)
		int3 localDimensions;
		int3 gridOffsets;

		// determine the direction of the plane (xy, xz or yz)
		if( direction.x == 1 ) {
			// yz plane (main direction: x)
			localDirection = make_int3( direction.y, direction.z, direction.x );
			localDimensions = make_int3( dimensions[1], dimensions[2], dimensions[0] );
			gridOffsets = make_int3(
					getDirectionOffset( yDirection ),
					getDirectionOffset( zDirection ),
					getDirectionOffset( xDirection )
				);
		}
		else if( direction.y == 1 ) {
			// xz plane (main direction: y)
			localDirection = make_int3( direction.x, direction.z, direction.y );
			localDimensions = make_int3( dimensions[0], dimensions[2], dimensions[1] );
			gridOffsets = make_int3(
					getDirectionOffset( xDirection ),
					getDirectionOffset( zDirection ),
					getDirectionOffset( yDirection )
				);
		}
		else if( direction.z == 1 ) {
			// xy plane (main direction: z)
			localDirection = direction;
			localDimensions = make_int3( dimensions[0], dimensions[1], dimensions[2] );
			gridOffsets = make_int3(
					getDirectionOffset( xDirection ),
					getDirectionOffset( yDirection ),
					getDirectionOffset( zDirection )
				);
		}
		else {
			assert( false );
		}

		// determine the startOffset as first cell near (0,0,0) so that start + neighborOffset won't be out of bounds
		int evenSlicesStartIndex = getCellOffset( -min( direction, zero3 ) );
		// odd slices start one slice "down"
		int oddSlicesStartIndex = evenSlicesStartIndex + gridOffsets.z;

		// set z to 0
		// adapt the local dimensions in such a way as to avoid out of bounds accesses at the "far corners"
		// the positive components of localSliceDirection affect the max corner of the slice
		// the negative ones the min corner (see *StartIndex). dimensions = max - min => use abs to count both correctly.
		localDimensions -= abs( localDirection );

		// always move 2 slices in local z direction, so we hit either odd or even slices in one kernel call
		gridOffsets.z *= 2;

		// there are floor( dimZ / 2 ) odd slices
		int numOddSlices = localDimensions.z / 2;
		int numEvenSlices = localDimensions.z - numOddSlices;

		int numCellsInSlice = localDimensions.x * localDimensions.y;

		// do all even slices
		Kernel_calculatePairLJForces<<<numEvenSlices * numCellsInSlice, blockSize>>>(
				_positions.devicePtr(), _forces.devicePtr(),_cellInfos.devicePtr(), _domainValues.devicePtr(),
				evenSlicesStartIndex, make_int2( localDimensions ), gridOffsets,
				neighborOffset,
				epsilon, sigmaSquared, cutOffRadiusSquared
			);

		// do all odd slices
		Kernel_calculatePairLJForces<<<numOddSlices * numCellsInSlice, blockSize>>>(
				_positions.devicePtr(), _forces.devicePtr(),_cellInfos.devicePtr(), _domainValues.devicePtr(),
				oddSlicesStartIndex, make_int2( localDimensions ), gridOffsets,
				neighborOffset,
				epsilon, sigmaSquared, cutOffRadiusSquared
			);
	}

	cellPairsTimer.end();

	singleCellsTimer.printElapsedTime( "intra cell LJ forces: %f ms " );
	cellPairsTimer.printElapsedTime( "inter cell LJ forces: %f ms\n" );
}

void LinkedCellsCUDA_Internal::reducePotentialAndVirial( OUT float &potential, OUT float &virial ) {
	const std::vector<unsigned long> &innerCellIndices = _linkedCells.getInnerCellIndices();
	const std::vector<unsigned long> &boundaryCellIndices = _linkedCells.getBoundaryCellIndices();

	potential = 0.0f;
	virial = 0.0f;
	for( int i = 0 ; i < innerCellIndices.size() ; i++ ) {
		int innerCellIndex = innerCellIndices[i];
#ifdef TEST_CELL_COVERAGE
		if( (int) _domainValues[ innerCellIndex ].x != 26 ) {
			printf( "%i (badly covered inner cell - coverage: %f)\n", innerCellIndex, _domainValues[ innerCellIndex ].x );
		}
#endif
		potential += _domainValues[ innerCellIndex ].x;
		virial += _domainValues[ innerCellIndex ].y;
	}
	for( int i = 0 ; i < boundaryCellIndices.size() ; i++ ) {
		int boundaryCellIndex = boundaryCellIndices[ i ];

#ifdef TEST_CELL_COVERAGE
		if( (int) _domainValues[ boundaryCellIndex ].x != 26 ) {
			printf( "%i (badly covered inner cell - coverage: %f)\n", boundaryCellIndex, _domainValues[ boundaryCellIndex ].x );
		}
#endif

		potential += _domainValues[ boundaryCellIndex ].x;
		virial += _domainValues[ boundaryCellIndex ].y;
	}

	// every contribution is added twice so divide by 2
	potential /= 2.0f;
	virial /= 2.0f;

	// TODO: I have no idea why the sign is different in the GPU code...
	virial = -virial;
}
