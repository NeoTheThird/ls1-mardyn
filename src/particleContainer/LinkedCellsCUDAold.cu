#include "hip/hip_runtime.h"
// Andreas Kirsch <kirschan@tum.de>

#include "LinkedCellsCUDA.h"
#include "LinkedCells.h"
#include "molecules/potforce.h"
#include "handlerInterfaces/ParticlePairsHandler.h"
#include "Cell.h"
#include "molecules/Molecule.h"
#include "ensemble/GrandCanonical.h"
#include "io/OneCLJGenerator.h"
#include <sys/time.h>

#include <string>

#include <hip/hip_runtime.h>
#include "cutil_math.h"

struct CUDAException : public std::exception {
	const hipError_t errorCode;
	const std::string errorSource;

	CUDAException( hipError_t errorCode, const std::string &errorSource = "" )
	: errorCode( errorCode ), errorSource( errorSource ) {}

	~CUDAException() throw() {}

    /** Returns a C-style character string describing the general cause
     *  of the current error.  */
    virtual const char* what() const throw() {
    	return errorSource.c_str();
    }
};

#define CUDA_THROW_ON_ERROR( expr ) \
	do { \
		hipError_t errorCode = (expr); \
		if( errorCode != hipSuccess ) { \
			throw CUDAException( errorCode, #expr ); \
		} \
	} while( 0 )


extern __shared__ float4 pblock[];
__global__ void LJ( float4* pos,			//positions of the particles
					int* pairs,			//the cell pairs
					float4* force,
					int numberOfPairs, int maxCellSize,float cutOffRadius,float eps24,float sig2
					)
{
	int nt = 64;						//nt threads are responsible for each cell pair
	int n = gridDim.x * blockDim.x; //get_global_size(0); 		//size of the index space
  	int gti = (blockIdx.x * blockDim.x + threadIdx.x) / nt;		//index of the cell pair of this thread (global id divided by number of threads per pair)
  	int ti = threadIdx.x;			//index of this thread inside its workgroup
  	int mi= threadIdx.x % nt;		//index of the particle inside the first cell for which this thread is responsible
  										//if the cell contains more than nt particles, then mi is used as an offset

	if(gti < numberOfPairs) {
		int cell1=pairs[gti*5];				//startindex of first cell
		int cell2=pairs[gti*5+1];			//startindex of second cell
		int cellSize1=pairs[gti*5+2];		//size of first cell
		int cellSize2=pairs[gti*5+3];		//size of second cell
		//int offset=pairs[gti*5+4];
		float4 p;							//stores the current particle position
		float4 f;							//stores the current force of the particle
		float virial=0.;

		//if the cell contains more than nt particles, each thread has to be responsible for several particles
		//this loop iterates over them
		for(int i1=0; i1 < ceil((float)cellSize1/(float)nt)+1; i1++) {
			int index1=i1*nt+mi;		//index of current particle inside the first cell
			if(index1 < cellSize1) {	//check if index1 is inside the first cell
				p=pos[cell1+index1];
			}
			f.x=f.y=f.z=f.w=0.0f;
			for(int i2=0; i2 < ceil((float)cellSize2/(float)nt)+1; i2++) {
				int index2=i2*nt;
				if(index2+mi < cellSize2) {
					pblock[ti] = pos[cell2+index2+mi];
				}
				__syncthreads();
					if(index1 < cellSize1) {
						for(int m=0; m < nt; m++) {		 //iterate over all cached particles
							if(index2+m < cellSize2) {
								float4 p2=pblock[ti-mi+m];
								//calculate distance
								float4 d = p2 - p;
								d.w = d.x*d.x +d.y*d.y + d.z*d.z;
								if(d.w < cutOffRadius && d.w != 0) {	// pairs are inside the cut of radius and not the same particle
									//calculate the LJ Potential and sum it up:
									float invdr2=1.f/d.w;
  									float lj6=sig2*invdr2; lj6=lj6*lj6*lj6;
  									float lj12=lj6*lj6;
  									float lj12m6=lj12-lj6;
  									float u6=eps24*lj12m6+f.w;
  									float fac=eps24*(lj12+lj12m6)*invdr2;
  									//sum up the forces
  									f=f+fac*d;
  									f.w=u6;
  									virial+=f.x*d.x+f.y*d.y+f.z*d.z;  //needed for virial in
								}
							}
						}
					}
				__syncthreads();
			}
			if(index1 < cellSize1) {
			    int i=(cell1+index1)*2;
				force[i]=f;
				force[i+1].x=virial;
			}
		}
	}
}

LinkedCellsOpenCL::LinkedCellsOpenCL(double bBoxMin[3], double bBoxMax[3], double cutoffRadius, double LJCutoffRadius,
	     double tersoffCutoffRadius, double cellsInCutoffRadius,
	     ParticlePairsHandler* partPairsHandler)
			: LinkedCells(bBoxMin,bBoxMax,cutoffRadius,LJCutoffRadius,tersoffCutoffRadius,cellsInCutoffRadius,partPairsHandler) {
	LinkedCellsOpenCL::numberOfParticles=numberOfParticles;

	totalTime=0;
	numberOfParticles = 0;
}

LinkedCellsOpenCL::~LinkedCellsOpenCL() {
	cout << "LinkedCellsOpenCL::deconstruct" << endl;
}

void LinkedCellsOpenCL::initCUDA() {
	cout << "Initializing CUDA" << endl;

	try {
		int	deviceCount;
		CUDA_THROW_ON_ERROR( hipGetDevice( &deviceCount ) );

		cout << deviceCount << " devices found" << endl;
		cout << "Using device 0" << endl;
		CUDA_THROW_ON_ERROR( hipSetDevice( 0 ) );

		// TODO: output more information using cudaGetDeviceProps
	}
	catch( const CUDAException &hipError_t ) {
		std::cerr << "ERROR: " << hipError_t.errorSource << "(" << hipError_t.errorCode << ")"
				<< std::endl;

		exit(EXIT_FAILURE);
	}
}

void LinkedCellsOpenCL::invertPairs() {
	for(int i=0; i < numberOfPairs;++i) {
		int swapId=pairs[i*5];
		int swapSize=pairs[i*5+2];
		pairs[i*5]=pairs[i*5+1];
		pairs[i*5+2]=pairs[i*5+3];
		pairs[i*5+1]=swapId;
		pairs[i*5+3]=swapSize;
	}
}

//traverse pairs an calculate LJ on GPU
void LinkedCellsOpenCL::traversePairs() {
	timeval start,end;
	int numberOfCells=_cells.size();
	cout << "LinkedCellsOpenCL::traversePairs()" << endl;
	if(numberOfParticles == 0) {
		initCUDA();
		cout << "LinkedCellsOpenCL::countPairs()" << endl;
		countPairs();
		cout << "LinkedCellsOpenCL::countPairs():" << numberOfPairs << "  numberOfCells " << numberOfCells<< endl;
	}
	numberOfParticles=countParticles();
	cout << "LinkedCellsOpenCL::countParticles in Cells " << numberOfParticles<< endl;
	m_positions = (float*) memalign(16, numberOfParticles * sizeof(float4));
	cout << "LinkedCellsOpenCL::traversePairs()" << endl;
	cout << "LinkedCellsOpenCL::traversePairsInit()" << endl;
	traversePairsInit();




	 int maxCellSize=getMaxCellSizeAndPositions();


	pairs = (int*) memalign(16, 2*numberOfPairs*5* sizeof(int));

	cout << "createPairs-" << endl;
	createPairs();
	cout << "createPairs+" << endl;
	for(int m=0; m < numberOfPairs; ++m) {
		//cout << "pair: cell1 " <<  pairs[m*5] <<" cell2 " <<  pairs[m*5+1]<<" cell1 size " <<  pairs[m*5+2]<<" cell2 size "  <<  pairs[m*5+3]<<" offset " <<  pairs[m*5+4] << endl;
	}

	cout << "numberOfPairs " << numberOfPairs << " resultSize " << resultSize << endl;

	forces = (float*)  memalign(16, 2*numberOfParticles  * sizeof(float4));


	cout << "LinkedCellsOpenCL::calculate distances" << endl;

	double dif;
	gettimeofday(&start,NULL);
	try {
		void *memPositions, *memPairs, *memForces;

		// TODO: move the device memory allocation and release into the init/deinit functions
		CUDA_THROW_ON_ERROR( hipMalloc( &memPositions, numberOfParticles * sizeof(float4) ) );
		CUDA_THROW_ON_ERROR( hipMalloc( &memPairs, numberOfPairs *10* sizeof(int) ) );
		CUDA_THROW_ON_ERROR( hipMalloc( &memForces, 2*numberOfParticles * sizeof(float4) ) );

		CUDA_THROW_ON_ERROR( hipMemcpy( memPositions, m_positions, numberOfParticles * sizeof(float4), hipMemcpyHostToDevice ) );
		CUDA_THROW_ON_ERROR( hipMemcpy( memPairs, pairs, numberOfPairs *10* sizeof(int), hipMemcpyHostToDevice ) );
		CUDA_THROW_ON_ERROR( hipMemcpy( memForces, forces, 2*numberOfParticles * sizeof(float4), hipMemcpyHostToDevice ) );

		cout << "LinkedCellsOpenCL maxCellSize " << maxCellSize<< endl;

		int gridSize = (numberOfPairs-1) / max_threads + 1;

		cout << "LinkedCellsOpenCL::gridSize:" << gridSize << endl;

		size_t sharedMemorySize = max_threads * sizeof(float4);
		// TODO: refactor *64..
		LJ<<<gridSize*64, max_threads, sharedMemorySize>>>( (float4*) memPositions, (int*) memPairs, (float4*) memForces, numberOfPairs, maxCellSize,
				(float) cutoffRadiusSquare, (float) 24, (float) 1);

		CUDA_THROW_ON_ERROR( hipMemcpy( forces, memForces, 2*numberOfParticles	* sizeof(float4), hipMemcpyDeviceToHost ) );

		CUDA_THROW_ON_ERROR( hipFree( memPairs ) );
		CUDA_THROW_ON_ERROR( hipFree( memForces ) );

		invertPairs();
		float* tempForces=forces;

		forces = (float*)  memalign(16, 2*numberOfParticles  * sizeof(float4));

		void *memPairs2, *memForces2;
		CUDA_THROW_ON_ERROR( hipMalloc( &memPairs2, numberOfPairs *10* sizeof(int) ) );
		CUDA_THROW_ON_ERROR( hipMalloc( &memForces2, 2*numberOfParticles * sizeof(float4) ) );

		CUDA_THROW_ON_ERROR( hipMemcpy( memPairs2, pairs, numberOfPairs *10* sizeof(int), hipMemcpyHostToDevice ) );
		CUDA_THROW_ON_ERROR( hipMemcpy( memForces2, forces, 2*numberOfParticles * sizeof(float4), hipMemcpyHostToDevice ) );

		// TODO: refactor *64..
		LJ<<<gridSize*64, max_threads, sharedMemorySize>>>( (float4*) memPositions, (int*) memPairs2, (float4*) memForces2, numberOfPairs, maxCellSize,
				(float) cutoffRadiusSquare, (float) 24, (float) 1);

		CUDA_THROW_ON_ERROR( hipMemcpy( forces, memForces2, 2*numberOfParticles * sizeof(float4), hipMemcpyDeviceToHost ) );

		CUDA_THROW_ON_ERROR( hipFree( memPairs2 ) );
		CUDA_THROW_ON_ERROR( hipFree( memForces2 ) );

		//sum up the forces of the two steps:
		for(int m=0; m < numberOfParticles*8; ++m) {
				forces[m]+=tempForces[m];
		}
		free(tempForces);
		CUDA_THROW_ON_ERROR( hipFree( memPositions ) );
	}
	catch( const CUDAException &hipError_t ) {
		std::cerr << "ERROR in LinkedCellsOpenCL::traversePairs(): " << hipError_t.errorSource << "(" << hipError_t.errorCode << ")"
				<< std::endl;

		exit(EXIT_FAILURE);
	}

	gettimeofday(&end,NULL);
	dif=((end.tv_sec - start.tv_sec) * 1000000) +(end.tv_usec - start.tv_usec);
	dif= dif / 1000000;
	totalTime+=dif;
	cout << "calculated distances in  " << dif << " seconds total:  " << totalTime << endl;

	cout << "LinkedCellsOpenCL::calculate pairs" << endl;

	//copy the forces to the molecules:
	int index=0;
	std::list<Molecule*>::iterator molIter1;
	double uPot=0;
	float virial=0;
	for(unsigned i = 0; i < _cells.size(); i++ ){
					Cell& currentCell = _cells[i];
					for( molIter1 = currentCell.getParticlePointers().begin(); molIter1 != currentCell.getParticlePointers().end(); molIter1++ ) {
						Molecule& molecule1 = **molIter1;
						// TODOAK: hack hack hack.. constness is cast away
						double* Fsite=(double*) molecule1.ljcenter_F(0);
						Fsite[0]=(double)forces[index*8];
						Fsite[1]=(double)forces[index*8+1];
						Fsite[2]=(double)forces[index*8+2];
						uPot=uPot+(double)forces[index*8+3];
						virial+=forces[index*8+4];
						index++;
					}
	}

	cout << "LinkedCellsOpenCL::finish" << endl;
	traversePairsFinish();

	free(pairs);
	free(distances);
	free(m_positions);
}

void LinkedCellsOpenCL::traversePairsInit() {

	LinkedCells::_particlePairsHandler->init();

	// loop over all cells
	vector<Cell>::iterator cellIter;
	std::list<Molecule*>::iterator molIter1;
	std::list<Molecule*>::iterator molIter2;
	for( cellIter = _cells.begin(); cellIter !=  _cells.end(); cellIter++ ) {
		for( molIter1 = cellIter->getParticlePointers().begin(); molIter1 != cellIter->getParticlePointers().end(); molIter1++ ) {
			double zero[] = {0,0,0};
			(*molIter1)->setF( zero );
			(*molIter1)->setM( zero );
		}
	}

	vector<unsigned long>::iterator cellIndexIter;
	vector<unsigned long>::iterator neighbourOffsetsIter;

	// sqare of the cutoff radius
	 cutoffRadiusSquare = _cutoffRadius * _cutoffRadius;
	 LJCutoffRadiusSquare = _LJCutoffRadius * _LJCutoffRadius;
	tersoffCutoffRadiusSquare = _tersoffCutoffRadius * _tersoffCutoffRadius;


	for( unsigned i = 0; i < _cells.size(); i++ )
	{
		Cell& currentCell = _cells[i];
		for( molIter1 = currentCell.getParticlePointers().begin();
				molIter1!=currentCell.getParticlePointers().end();
				molIter1++ )
		{
			Molecule& molecule1 = **molIter1;
			molecule1.clearTersoffNeighbourList();
		}
	}
}

void LinkedCellsOpenCL::countPairs() {

	numberOfPairs=0;
	numberOfSelfs=0;




	vector<unsigned long>::iterator cellIndexIter;
	vector<unsigned long>::iterator neighbourOffsetsIter;
	std::list<Molecule*>::iterator molIter1;
		// loop over all inner cells and calculate forces to forward neighbours
		for( cellIndexIter = _innerCellIndices.begin(); cellIndexIter != _innerCellIndices.end(); cellIndexIter++ ) {
			Cell& currentCell = _cells[*cellIndexIter];
			numberOfSelfs++;
			numberOfPairs++;
			// loop over all neighbours
			for( neighbourOffsetsIter = _forwardNeighbourOffsets.begin(); neighbourOffsetsIter != _forwardNeighbourOffsets.end(); neighbourOffsetsIter++ )
			{
				Cell& neighbourCell = _cells[*cellIndexIter + *neighbourOffsetsIter];
				if(neighbourCell.getParticlePointers().size() != 0)numberOfPairs++;
			}
		}


		// loop over all boundary cells and calculate forces to forward and backward neighbours
		for( cellIndexIter = _boundaryCellIndices.begin(); cellIndexIter != _boundaryCellIndices.end(); cellIndexIter++ )
		{
			Cell& currentCell = _cells[*cellIndexIter];
			numberOfSelfs++;
			numberOfPairs++;
			// loop over all forward neighbours
			for( neighbourOffsetsIter = _forwardNeighbourOffsets.begin(); neighbourOffsetsIter != _forwardNeighbourOffsets.end(); neighbourOffsetsIter++ )
			{
				Cell& neighbourCell = _cells[*cellIndexIter + *neighbourOffsetsIter];
				numberOfPairs++;
			}

			// loop over all backward neighbours. calculate only forces
			// to neighbour cells in the halo region, all others already have been calculated
			for( neighbourOffsetsIter = _backwardNeighbourOffsets.begin(); neighbourOffsetsIter != _backwardNeighbourOffsets.end(); neighbourOffsetsIter++ )
			{
				Cell& neighbourCell = _cells[*cellIndexIter+*neighbourOffsetsIter];
				if(neighbourCell.isHaloCell())
				{
					numberOfPairs++;
				}
			}
		}

}
void LinkedCellsOpenCL::createPairs() {
	resultSize=0;
	seflResultSize=0;
	int pi=0;
	vector<unsigned long>::iterator cellIndexIter;
	vector<unsigned long>::iterator neighbourOffsetsIter;
	std::list<Molecule*>::iterator molIter1;
		// loop over all inner cells and calculate forces to forward neighbours
		for( cellIndexIter = _innerCellIndices.begin(); cellIndexIter != _innerCellIndices.end(); cellIndexIter++ ) {
			Cell& currentCell = _cells[*cellIndexIter];
			pairs[pi++]=currentCell.id();
			pairs[pi++]=currentCell.id();
			pairs[pi++]=currentCell.getParticlePointers().size();
			pairs[pi++]=currentCell.getParticlePointers().size();
			pairs[pi++]=resultSize;
			resultSize+=currentCell.getParticlePointers().size()*currentCell.getParticlePointers().size();

			seflResultSize+=currentCell.getParticlePointers().size();//*currentCell.getParticlePointers().size();

			// loop over all neighbours
			for( neighbourOffsetsIter = _forwardNeighbourOffsets.begin(); neighbourOffsetsIter != _forwardNeighbourOffsets.end(); neighbourOffsetsIter++ )
			{
				Cell& neighbourCell = _cells[*cellIndexIter + *neighbourOffsetsIter];
				if(neighbourCell.getParticlePointers().size() != 0) {
				pairs[pi++]=currentCell.id();
				pairs[pi++]=neighbourCell.id();
				pairs[pi++]=currentCell.getParticlePointers().size();
				pairs[pi++]=neighbourCell.getParticlePointers().size();
				pairs[pi++]=resultSize;
				resultSize+=currentCell.getParticlePointers().size()*neighbourCell.getParticlePointers().size();
				}
			}
		}


		// loop over all boundary cells and calculate forces to forward and backward neighbours
		for( cellIndexIter = _boundaryCellIndices.begin(); cellIndexIter != _boundaryCellIndices.end(); cellIndexIter++ )
		{
			Cell& currentCell = _cells[*cellIndexIter];
			pairs[pi++]=currentCell.id();
			pairs[pi++]=currentCell.id();
			pairs[pi++]=currentCell.getParticlePointers().size();
			pairs[pi++]=currentCell.getParticlePointers().size();
			pairs[pi++]=resultSize;
			resultSize+=currentCell.getParticlePointers().size()*currentCell.getParticlePointers().size();

			seflResultSize+=currentCell.getParticlePointers().size();//*currentCell.getParticlePointers().size();
			// loop over all forward neighbours
			for( neighbourOffsetsIter = _forwardNeighbourOffsets.begin(); neighbourOffsetsIter != _forwardNeighbourOffsets.end(); neighbourOffsetsIter++ )
			{
				Cell& neighbourCell = _cells[*cellIndexIter + *neighbourOffsetsIter];
				pairs[pi++]=currentCell.id();
				pairs[pi++]=neighbourCell.id();
				pairs[pi++]=currentCell.getParticlePointers().size();
				pairs[pi++]=neighbourCell.getParticlePointers().size();
				pairs[pi++]=resultSize;
				resultSize+=currentCell.getParticlePointers().size()*neighbourCell.getParticlePointers().size();
			}
			// loop over all backward neighbours. calculate only forces
			// to neighbour cells in the halo region, all others already have been calculated
			for( neighbourOffsetsIter = _backwardNeighbourOffsets.begin(); neighbourOffsetsIter != _backwardNeighbourOffsets.end(); neighbourOffsetsIter++ )
			{
				Cell& neighbourCell = _cells[*cellIndexIter+*neighbourOffsetsIter];
				if(neighbourCell.isHaloCell())
				{
				 pairs[pi++]=currentCell.id();
				 pairs[pi++]=neighbourCell.id();
				 pairs[pi++]=currentCell.getParticlePointers().size();
				 pairs[pi++]=neighbourCell.getParticlePointers().size();
				 pairs[pi++]=resultSize;
				 resultSize+=currentCell.getParticlePointers().size()*neighbourCell.getParticlePointers().size();
				}
			}
		}
}

void LinkedCellsOpenCL::traversePairsFinish() {
	double params[15];
		double delta_r;
		bool knowparams = false;
		std::list<Molecule*>::iterator molIter1;
		vector<unsigned long>::iterator cellIndexIter;
		for( cellIndexIter = _innerCellIndices.begin(); cellIndexIter != _boundaryCellIndices.end(); cellIndexIter++ )
		{
			if( cellIndexIter == _innerCellIndices.end() )
				cellIndexIter = _boundaryCellIndices.begin();
			Cell& currentCell = _cells[*cellIndexIter];
			for( molIter1 = currentCell.getParticlePointers().begin(); molIter1 != currentCell.getParticlePointers().end(); molIter1++ )
			{
				Molecule& molecule1 = **molIter1;
				if( molecule1.numTersoff() == 0 ) continue;
				if( !knowparams )
				{
					delta_r = molecule1.tersoffParameters(params);
					knowparams = true;
				}
				_particlePairsHandler->processTersoffAtom(molecule1, params, delta_r);
			}
		}

		_particlePairsHandler->finish();
}

int LinkedCellsOpenCL::countParticles() {
	int result=0;
	std::list<Molecule*>::iterator molIter1;
	for(unsigned i = 0; i < _cells.size(); i++ ){
		result+=_cells[i].getParticlePointers().size();
	}
	return result;
}

int LinkedCellsOpenCL::getMaxCellSizeAndPositions() {
	int maxSize=0;
	int index=0;
	int offset=0;
	std::list<Molecule*>::iterator molIter1;
	for(unsigned i = 0; i < _cells.size(); i++ ){
			Cell& currentCell = _cells[i];
			if (_cellsValid == false) {
					cout << "Cell structure in LinkedCells (traversePairs) invalid, call update first" << endl;
					exit(1);
				}

			currentCell.setId(offset);
			offset+=currentCell.getParticlePointers().size();
			//if(currentCell.getParticlePointers().size() != 0)cout << "cell size: " << currentCell.getParticlePointers().size()<< endl;
			maxSize=max(maxSize,(int)currentCell.getParticlePointers().size());
			if( currentCell.getParticlePointers().size() == 0 ) {
				continue;
			}
			for( molIter1 = currentCell.getParticlePointers().begin(); molIter1 != currentCell.getParticlePointers().end(); molIter1++ ) {
				Molecule& molecule1 = **molIter1;
				if(index >= numberOfParticles) {
					cout << "found more particles then declared  "<< index<< endl;
					//exit(1);
				} else {
					m_positions[index*4]=molecule1.r(0);
					m_positions[index*4+1]=molecule1.r(1);
					m_positions[index*4+2]=molecule1.r(2);
					m_positions[index*4+3]=index;
				}
				index++;

			}
	}
	return maxSize;
}
