#include "hip/hip_runtime.h"
#include <host_defines.h>
#include <stdio.h>

#include "cutil_math.h"

#include "config.h"

__device__ int getSM() {
	uint smID;

	asm volatile( "mov.u32 %0, %smid;" : "=r"(smID) );

	return smID;
}

#define warpThreadIdx threadIdx.x
#define warpIdx threadIdx.y

__device__ __forceinline__ uint getThreadIndex() {
	return warpThreadIdx + WARP_SIZE * warpIdx;
}

// use these printf wrappers for simplified debug output
#if 0
#	define THREAD_PRINTF(format, ...) printf( "({%i} %i W%i T%i) " format, blockIdx.x + blockIdx.y * gridDim.x, warpIdx, warpThreadIdx, getSM(), ##__VA_ARGS__ )
#	define WARP_PRINTF(format, ...) do { if( warpThreadIdx == 0 ) printf( "({%i} %i W%i) " format, blockIdx.x + blockIdx.y * gridDim.x, warpIdx, getSM(), ##__VA_ARGS__ ); } while( false )
#	define BLOCK_PRINTF(format, ...) do { if( warpIdx == 0 ) printf( "({%i} %i) " format, blockIdx.x + blockIdx.y * gridDim.x, getSM(), ##__VA_ARGS__ ); } while( false )
#	define GRID_PRINTF(format, ...) do { if( blockIdx.x == 0 && blockIdx.y == 0 && warpThreadIdx == 0 && warpIdx == 0 ) printf( "{%i} " format, getSM(), ##__VA_ARGS__ ); } while( false )
#else
#	define THREAD_PRINTF(format, ...)
#	define WARP_PRINTF(format, ...)
#	define BLOCK_PRINTF(format, ...)
#	define GRID_PRINTF(format, ...)
#endif


#include "domainTraverser.cum"
#include "globalStats.cum"

#include "referenceCellProcessor.cum"
#include "threadBlockCellProcessor.cum"
#include "warpBlockCellProcessor.cum"

#ifdef REFERENCE_IMPLEMENTATION
#warning using reference cell processor
#endif

#ifdef CUDA_DOUBLE_MODE
#	warning using double precision
#else
#	warning using float precision
#endif

extern "C" {
/* TODO: possible refactoring
 * create a prepare method in MoleculeStorage
 */
// TODO: interesting to benchmark idea: unrolled loop in this kernel vs the way it is now---overhead?
__global__ void convertQuaternionsToRotations( int numMolecules ) {
	int moleculeIndex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if( moleculeIndex >= numMolecules ) {
		return;
	}

	const Quaternion quaternion = moleculeQuaternions[ moleculeIndex ];

#ifndef TEST_QUATERNION_MATRIX_CONVERSION
	moleculeRotations[ moleculeIndex ] = quaternion.toRotMatrix3x3();
#else
#	warning CUDA: testing quaternion matrix conversion
	const Matrix3x3 convertedQuaternion = quaternion.toRotMatrix3x3();
	const Matrix3x3 &correctRotation = moleculeRotations[ moleculeIndex ];

	const floatType error = length( convertedQuaternion.rows[0] - correctRotation.rows[0] ) +
			length( convertedQuaternion.rows[1] - correctRotation.rows[1] ) +
			length( convertedQuaternion.rows[2] - correctRotation.rows[2] );

	if( error > 1e-9 ) {
		printf( "bad quaternion conversion (molecule %i)\n", moleculeIndex );
	}
#endif
}

#ifndef CUDA_WARP_BLOCK_CELL_PROCESSOR

#ifndef REFERENCE_IMPLEMENTATION
namespace CellProcessor = ThreadBlockCellProcessor;
#else
namespace CellProcessor = ReferenceCellProcessor;
#endif

__global__ void processCellPair() {
	const int threadIndex = getThreadIndex();

	const int jobIndex = blockIdx.y * gridDim.x + blockIdx.x;
	if( jobIndex >= DomainTraverser::numJobs ) {
		return;
	}

	int cellIndex = DomainTraverser::getCellIndexFromJobIndex( jobIndex );
	int neighborIndex = DomainTraverser::getNeighborCellIndex( cellIndex );

	CellInfoEx cellA = cellInfoFromCellIndex( cellIndex );
	CellInfoEx cellB = cellInfoFromCellIndex( neighborIndex );

	if( cellA.length == 0 || cellB.length == 0 ) {
		return;
	}

	ThreadBlockCellStats::initThreadLocal( threadIndex );
	CellProcessor::processCellPair( threadIndex, cellA, cellB );

	ThreadBlockCellStats::reduceAndStore( threadIndex, cellIndex, neighborIndex );
}

__global__ void processCell() {
	const int threadIndex = getThreadIndex();

	int jobIndex = blockIdx.y * gridDim.x + blockIdx.x;
	if( jobIndex >= DomainTraverser::numJobs ) {
		return;
	}

	int cellIndex = DomainTraverser::getInnerCellIndexFromJobIndex(jobIndex);
	CellInfoEx cell = cellInfoFromCellIndex( cellIndex );
	if( cell.length == 0 ) {
		return;
	}

	ThreadBlockCellStats::initThreadLocal( threadIndex );
	CellProcessor::processCell( threadIndex, cell );

	ThreadBlockCellStats::reduceAndStore( threadIndex, cellIndex, cellIndex );
}
#else
namespace CellProcessor = WarpBlockCellProcessor;

__device__ CellProcessor::CellScheduler *cellScheduler;
__device__ CellProcessor::CellPairScheduler *cellPairScheduler;

__global__ void createSchedulers() {
	cellScheduler = new CellProcessor::CellScheduler();
	cellPairScheduler = new CellProcessor::CellPairScheduler();
}

__global__ void destroySchedulers() {
	delete cellScheduler;
	delete cellPairScheduler;
}

__global__ void processCellPair() {
	const int threadIndex = getThreadIndex();

	__shared__ CellProcessor::ThreadBlockInfo threadBlockInfo;
	if( threadIndex == 0 ) {
		threadBlockInfo.init();
	}
	__syncthreads();

	do {
		cellPairScheduler->scheduleWarpBlocks( threadBlockInfo );

		while( !threadBlockInfo.warpJobQueue[warpIdx].isEmpty() ) {
			ThreadBlockCellStats::initThreadLocal( threadIndex );

			CellProcessor::WarpBlockPairInfo warpBlockPairInfo = threadBlockInfo.warpJobQueue[warpIdx].pop();
			CellProcessor::processCellPair( warpBlockPairInfo );

			ThreadBlockCellStats::reduceAndStoreWarp( threadIndex, warpBlockPairInfo.warpBlockA.cellIndex );
		}
	} while( threadBlockInfo.hasMoreJobs );

	WARP_PRINTF( "terminating\n" );
}

__global__ void processCell() {
	// TODO: remove?
	const int threadIndex = getThreadIndex();

	__shared__ CellProcessor::ThreadBlockInfo threadBlockInfo;
	if( threadIndex == 0 ) {
		threadBlockInfo.init();
	}
	__syncthreads();

	do {
		cellScheduler->scheduleWarpBlocks( threadBlockInfo );

		while( !threadBlockInfo.warpJobQueue[warpIdx].isEmpty() ) {
			ThreadBlockCellStats::initThreadLocal( threadIndex );

			CellProcessor::WarpBlockPairInfo warpBlockPairInfo = threadBlockInfo.warpJobQueue[warpIdx].pop();
			CellProcessor::processCell( warpBlockPairInfo );

			ThreadBlockCellStats::reduceAndStoreWarp( threadIndex, warpBlockPairInfo.warpBlockA.cellIndex );
		}
	} while( threadBlockInfo.hasMoreJobs );

	WARP_PRINTF( "terminating\n" );
}

#endif

}
