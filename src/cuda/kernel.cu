#include "hip/hip_runtime.h"
#include <host_defines.h>

__device__ int getSM() {
	uint smID;

	asm volatile( "mov.u32 %0, %smid;" : "=r"(smID) );

	return smID;
}

#if 0
#	define WARP_PRINTF(format, ...) if( threadIdx.x == 0 ) printf( "(%i W%i {%i}) " format, blockIdx.x + blockIdx.y * gridDim.x, threadIdx.y, getSM(), ##__VA_ARGS__ )
#	define BLOCK_PRINTF(format, ...) if( threadIdx.y == 0 ) printf( "(%i {%i}) " format, blockIdx.x + blockIdx.y * gridDim.x, getSM(), ##__VA_ARGS__ )
#	define GRID_PRINTF(format, ...) if( blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0 ) printf( "{%i} " format, getSM(), ##__VA_ARGS__ )
#else
#	define WARP_PRINTF(format, ...)
#	define BLOCK_PRINTF(format, ...)
#	define GRID_PRINTF(format, ...)
#endif

#include <stdio.h>

#include "cutil_math.h"

#include "moleculeStorage.cum"

#include "componentDescriptor.cum"

#include "moleculePairHandler.cum"

#include "domainTraverser.cum"

#include "domainProcessor.cum"

#include "globalStats.cum"

#include "cellInfo.cum"

#include "molecule.cum"

#include "warpBlockDomainProcessor.cum"

#include "config.h"

#ifndef REFERENCE_IMPLEMENTATION
#warning using fast cell processor
#else
#warning using reference cell processor
#endif

#ifdef CUDA_DOUBLE_MODE
#	warning using double precision
#else
#	warning using float precision
#endif

#ifdef CUDA_SORT_CELLS_BY_COMPONENTTYPE
#	warning sorting cells by component type
#else
#	warning cells are *not* sorted by component type
#endif

#ifdef CUDA_HW_CACHE_ONLY
#	warning no shared local storage cache
#else
#	warning shared local storage active
#endif

#ifdef NO_CONSTANT_MEMORY
#	warning no constant memory
#else
#	warning constant memory used
#endif

extern "C" {
/* TODO: possible refactoring
 * create a prepare method in MoleculeStorage
 */
// TODO: interesting to benchmark idea: unrolled loop in this kernel vs the way it is now---overhead?
__global__ void convertQuaternionsToRotations( int numMolecules ) {
	int moleculeIndex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if( moleculeIndex >= numMolecules ) {
		return;
	}

#ifndef CUDA_UNPACKED_STORAGE
	const Quaternion quaternion = moleculeQuaternions[ moleculeIndex ];
#else
	const Quaternion quaternion = packQuaternion( moleculeQuaternions, moleculeIndex );
#endif

#ifndef TEST_QUATERNION_MATRIX_CONVERSION
#	ifndef CUDA_UNPACKED_STORAGE
	moleculeRotations[ moleculeIndex ] = quaternion.toRotMatrix3x3();
#	else
	unpackMatrix3x3( moleculeRotations, moleculeIndex, quaternion.toRotMatrix3x3() );
#	endif
#else
#	warning CUDA: testing quaternion matrix conversion
	const Matrix3x3 convertedQuaternion = quaternion.toRotMatrix3x3();
	const Matrix3x3 &correctRotation = moleculeRotations[ moleculeIndex ];

	const floatType error = length( convertedQuaternion.rows[0] - correctRotation.rows[0] ) +
			length( convertedQuaternion.rows[1] - correctRotation.rows[1] ) +
			length( convertedQuaternion.rows[2] - correctRotation.rows[2] );

	if( error > 1e-9 ) {
		printf( "bad quaternion conversion (molecule %i)\n", moleculeIndex );
	}
#endif
}

#ifndef CUDA_WARP_BLOCK_CELL_PROCESSOR

__device__ MoleculeStorage moleculeStorage;

#ifndef REFERENCE_IMPLEMENTATION
#	ifndef CUDA_HW_CACHE_ONLY
__shared__ SharedMoleculeLocalStorage< typeof(moleculeStorage), moleculeStorage> moleculeLocalStorage;
#	else
__device__ WriteThroughMoleculeLocalStorage<typeof(moleculeStorage), moleculeStorage> moleculeLocalStorage;
#	endif
__device__ HighDensityDomainProcessor<
	typeof(moleculeStorage), moleculeStorage,
	typeof(moleculeLocalStorage), moleculeLocalStorage>
		domainProcessor;

#else
__device__ ReferenceCellProcessor<
	typeof(moleculeStorage), moleculeStorage,
	typeof(moleculePairHandler), moleculePairHandler>
		domainProcessor;
#endif

__global__ void processCellPair() {
	const int threadIndex = threadIdx.y * WARP_SIZE + threadIdx.x;

	const int jobIndex = blockIdx.y * gridDim.x + blockIdx.x;
	if( jobIndex >= DomainTraverser::numJobs ) {
		return;
	}

	int cellIndex = DomainTraverser::getCellIndexFromJobIndex( jobIndex );
	int neighborIndex = DomainTraverser::getNeighborCellIndex( cellIndex );

	CellInfoEx cellA = cellInfoFromCellIndex( cellIndex );
	CellInfoEx cellB = cellInfoFromCellIndex( neighborIndex );

	if( cellA.length == 0 || cellB.length == 0 ) {
		return;
	}

	ThreadBlockCellStats::initThreadLocal( threadIndex );
	domainProcessor.processCellPair( threadIndex, cellA, cellB );

	ThreadBlockCellStats::reduceAndStore( threadIndex, cellIndex, neighborIndex );
}

//__launch_bounds__(BLOCK_SIZE, 2)
__global__ void processCell() {
	const int threadIndex = threadIdx.y * WARP_SIZE + threadIdx.x;

	int jobIndex = blockIdx.y * gridDim.x + blockIdx.x;
	if( jobIndex >= DomainTraverser::numJobs ) {
		return;
	}

	int cellIndex = DomainTraverser::getInnerCellIndexFromJobIndex(jobIndex);
	CellInfoEx cell = cellInfoFromCellIndex( cellIndex );
	if( cell.length == 0 ) {
		return;
	}

	ThreadBlockCellStats::initThreadLocal( threadIndex );
	domainProcessor.processCell( threadIndex, cell );

	ThreadBlockCellStats::reduceAndStore( threadIndex, cellIndex, cellIndex );
}
#else

__device__ WBDP::CellScheduler *cellScheduler;
__device__ WBDP::CellPairScheduler *cellPairScheduler;

__global__ void createSchedulers() {
	if( threadIdx.x + threadIdx.y == 0 ) {
		cellScheduler = new WBDP::CellScheduler();
		cellPairScheduler = new WBDP::CellPairScheduler();
	}
}

__global__ void destroySchedulers() {
	if( threadIdx.x + threadIdx.y == 0 ) {
		delete cellScheduler;
		delete cellPairScheduler;
	}
}

__device__ MoleculeStorage moleculeStorage;
__device__ WBDP::DomainProcessor<typeof(moleculeStorage), moleculeStorage> domainProcessor;

__global__ void processCellPair() {
	const int threadIndex = threadIdx.y * WARP_SIZE + threadIdx.x;

	__shared__ WBDP::ThreadBlockInfo threadBlockInfo;

	do {
		cellPairScheduler->scheduleWarpBlocks( threadBlockInfo );

		while( !threadBlockInfo.warpJobQueue[threadIdx.y].isEmpty() ) {
			ThreadBlockCellStats::initThreadLocal( threadIndex );

			WBDP::WarpBlockPairInfo warpBlockPairInfo = threadBlockInfo.warpJobQueue[threadIdx.y].pop();
			domainProcessor.processCellPair( warpBlockPairInfo );

			ThreadBlockCellStats::reduceAndStoreWarp( threadIndex, warpBlockPairInfo.warpBlockA.cellIndex );
		}
	} while( threadBlockInfo.hasMoreJobs );

	WARP_PRINTF( "terminating..\n" );
}

__global__ void processCell() {
	// TODO: remove?
	const int threadIndex = threadIdx.y * WARP_SIZE + threadIdx.x;

	__shared__ WBDP::ThreadBlockInfo threadBlockInfo;

	do {
		cellScheduler->scheduleWarpBlocks( threadBlockInfo );

		while( !threadBlockInfo.warpJobQueue[threadIdx.y].isEmpty() ) {
			ThreadBlockCellStats::initThreadLocal( threadIndex );

			WBDP::WarpBlockPairInfo warpBlockPairInfo = threadBlockInfo.warpJobQueue[threadIdx.y].pop();
			domainProcessor.processCellPair( warpBlockPairInfo );

			ThreadBlockCellStats::reduceAndStoreWarp( threadIndex, warpBlockPairInfo.warpBlockA.cellIndex );
		}
	} while( threadBlockInfo.hasMoreJobs );

	WARP_PRINTF( "terminating\n" );
}

#endif

}
