#include "hip/hip_runtime.h"
#include "stdio.h"

#include "cutil_math.h"

#include "moleculeStorage.cum"

#include "componentDescriptor.cum"

#include "pairTraverser.cum"

#include "cellProcessor.cum"

#include "globalStats.cum"

#include "cellInfo.cum"

#include "molecule.cum"

#include "potForce.cum"

#include "moleculePairHandler.cum"

#include "config.h"

#ifndef REFERENCE_IMPLEMENTATION
#warning using fast cell processor
#else
#warning using reference cell processor
#endif

extern "C" {
/* TODO: possible refactoring
 * create a prepare method in MoleculeStorage and make rawQuaternions a global pointer
 * and forward the kernel call to it
 */
// TODO: interesting to benchmark: unrolled loop in this kernel vs the way it is now---overhead?
__global__ void convertQuaternionsToRotations( const QuaternionStorage *rawQuaternions, int numMolecules ) {
	const Quaternion *quaternions = (Quaternion*) rawQuaternions;

	int moleculeIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if( moleculeIndex < numMolecules ) {
#ifndef TEST_QUATERNION_MATRIX_CONVERSION
		moleculeRotations[ moleculeIndex ] = quaternions[ moleculeIndex ].toInvRotMatrix3x3();
#else
#warning CUDA: testing quaternion matrix conversion
		const Matrix3x3 convertedQuaternion = quaternions[ moleculeIndex ].toInvRotMatrix3x3();
		const Matrix3x3 &correctRotation = moleculeRotations[ moleculeIndex ];

		const float error = length( convertedQuaternion.rows[0] - correctRotation.rows[0] ) +
				length( convertedQuaternion.rows[1] - correctRotation.rows[1] ) +
				length( convertedQuaternion.rows[2] - correctRotation.rows[2] );

		if( error > 1e-9 ) {
			printf( "bad quaternion conversion (molecule %i)\n", moleculeIndex );
		}
#endif
	}
}

__global__ void processCellPair( int startIndex, int2 dimension, int3 gridOffsets, int neighborOffset ) {
	const int threadIndex = threadIdx.y * warpSize + threadIdx.x;

	int cellIndex = getCellIndex( startIndex, dimension, gridOffsets );
	int neighborIndex = cellIndex + neighborOffset;

	// TODO: move the swapping bit into the cell processor!
	/*int cellLength = cellInfos[ cellIndex + 1 ] - cellInfos[ cellIndex ];
	int neighborLength = cellInfos[ neighborIndex + 1 ] - cellInfos[ neighborIndex ];

	// ensure that cellA_length <= cellB_length (which will use fewer data transfers)
	// (numTransfersA + numTransfersA * numTransfersB) * BLOCK_SIZE
	if( cellLength > neighborLength ) {
		// swap cellIndex and neighborIndex
		cellIndex = neighborIndex;
		neighborIndex -= neighborOffset;
	}*/

	__shared__ CellStatsCollector<BLOCK_SIZE> globalStatsCollector;
	globalStatsCollector.initThreadLocal( threadIndex );

	ComponentDescriptorAccessor componentDescriptorAccessor;

	MoleculeStorage moleculeStorage;

	MoleculePairHandler<typeof(globalStatsCollector), typeof(componentDescriptorAccessor)> moleculePairHandler( globalStatsCollector, componentDescriptorAccessor );

#ifndef REFERENCE_IMPLEMENTATION
	FastCellProcessor<BLOCK_SIZE, Molecule, typeof(moleculeStorage), typeof(moleculePairHandler)> cellProcessor(moleculeStorage, moleculePairHandler);
#else
	ReferenceCellProcessor<Molecule, typeof(moleculeStorage), typeof(moleculePairHandler)> cellProcessor(moleculeStorage, moleculePairHandler);
#endif

	cellProcessor.processCellPair( threadIndex, cellInfoFromCellIndex( cellIndex ), cellInfoFromCellIndex( neighborIndex ) );

	globalStatsCollector.reduceAndSave( threadIndex, cellIndex, neighborIndex );
}

__global__ void processCell() {
	const int threadIndex = threadIdx.y * warpSize + threadIdx.x;

	int cellIndex = blockIdx.x;

	__shared__ CellStatsCollector<BLOCK_SIZE> globalStatsCollector;
	globalStatsCollector.initThreadLocal( threadIndex );

	ComponentDescriptorAccessor componentDescriptorAccessor;

	MoleculeStorage moleculeStorage;

	MoleculePairHandler<typeof(globalStatsCollector), typeof(componentDescriptorAccessor)> moleculePairHandler( globalStatsCollector, componentDescriptorAccessor );

#ifndef REFERENCE_IMPLEMENTATION
	FastCellProcessor<BLOCK_SIZE, Molecule, typeof(moleculeStorage), typeof(moleculePairHandler)> cellProcessor(moleculeStorage, moleculePairHandler);
#else
	ReferenceCellProcessor<Molecule, typeof(moleculeStorage), typeof(moleculePairHandler)> cellProcessor(moleculeStorage, moleculePairHandler);
#endif

	cellProcessor.processCell( threadIndex, cellInfoFromCellIndex( cellIndex ) );

	globalStatsCollector.reduceAndSave( threadIndex, cellIndex, cellIndex );
}
}
