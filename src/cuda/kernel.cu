#include "hip/hip_runtime.h"
#include <host_defines.h>
#include <stdio.h>

#include "cutil_math.h"

#include "config.h"

__device__ int getSM() {
	uint smID;

	asm volatile( "mov.u32 %0, %smid;" : "=r"(smID) );

	return smID;
}

#define warpThreadIdx threadIdx.x
#define warpIdx threadIdx.y

__device__ __forceinline__ uint getThreadIndex() {
	return warpThreadIdx + WARP_SIZE * warpIdx;
}

#if 0
#	define WARP_PRINTF(format, ...) if( warpThreadIdx == 0 ) printf( "(%i W%i {%i}) " format, blockIdx.x + blockIdx.y * gridDim.x, warpIdx, getSM(), ##__VA_ARGS__ )
#	define BLOCK_PRINTF(format, ...) if( warpIdx == 0 ) printf( "(%i {%i}) " format, blockIdx.x + blockIdx.y * gridDim.x, getSM(), ##__VA_ARGS__ )
#	define GRID_PRINTF(format, ...) if( blockIdx.x == 0 && blockIdx.y == 0 && warpThreadIdx == 0 && warpIdx == 0 ) printf( "{%i} " format, getSM(), ##__VA_ARGS__ )
#else
#	define WARP_PRINTF(format, ...)
#	define BLOCK_PRINTF(format, ...)
#	define GRID_PRINTF(format, ...)
#endif

#include "moleculeStorage.cum"

#include "componentDescriptor.cum"

#include "moleculePairHandler.cum"

#include "domainTraverser.cum"

#include "referenceCellProcessor.cum"

#include "threadBlockCellProcessor.cum"

#include "globalStats.cum"

#include "cellInfo.cum"

#include "molecule.cum"

#include "warpBlockCellProcessor.cum"

#ifndef REFERENCE_IMPLEMENTATION
#warning using fast cell processor
#else
#warning using reference cell processor
#endif

#ifdef CUDA_DOUBLE_MODE
#	warning using double precision
#else
#	warning using float precision
#endif

#ifdef CUDA_SORT_CELLS_BY_COMPONENTTYPE
#	warning sorting cells by component type
#else
#	warning cells are *not* sorted by component type
#endif

#ifdef CUDA_HW_CACHE_ONLY
#	warning no shared local storage cache
#else
#	warning shared local storage active
#endif

#ifdef NO_CONSTANT_MEMORY
#	warning no constant memory
#else
#	warning constant memory used
#endif

extern "C" {
/* TODO: possible refactoring
 * create a prepare method in MoleculeStorage
 */
// TODO: interesting to benchmark idea: unrolled loop in this kernel vs the way it is now---overhead?
__global__ void convertQuaternionsToRotations( int numMolecules ) {
	int moleculeIndex = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if( moleculeIndex >= numMolecules ) {
		return;
	}

#ifndef CUDA_UNPACKED_STORAGE
	const Quaternion quaternion = moleculeQuaternions[ moleculeIndex ];
#else
	const Quaternion quaternion = packQuaternion( moleculeQuaternions, moleculeIndex );
#endif

#ifndef TEST_QUATERNION_MATRIX_CONVERSION
#	ifndef CUDA_UNPACKED_STORAGE
	moleculeRotations[ moleculeIndex ] = quaternion.toRotMatrix3x3();
#	else
	unpackMatrix3x3( moleculeRotations, moleculeIndex, quaternion.toRotMatrix3x3() );
#	endif
#else
#	warning CUDA: testing quaternion matrix conversion
	const Matrix3x3 convertedQuaternion = quaternion.toRotMatrix3x3();
	const Matrix3x3 &correctRotation = moleculeRotations[ moleculeIndex ];

	const floatType error = length( convertedQuaternion.rows[0] - correctRotation.rows[0] ) +
			length( convertedQuaternion.rows[1] - correctRotation.rows[1] ) +
			length( convertedQuaternion.rows[2] - correctRotation.rows[2] );

	if( error > 1e-9 ) {
		printf( "bad quaternion conversion (molecule %i)\n", moleculeIndex );
	}
#endif
}

#ifndef CUDA_WARP_BLOCK_CELL_PROCESSOR

__device__ MoleculeStorage moleculeStorage;

#ifndef REFERENCE_IMPLEMENTATION
#	ifndef CUDA_HW_CACHE_ONLY
__shared__ SharedMoleculeLocalStorage< moleculeStorage > moleculeLocalStorage;
#	else
__device__ WriteThroughMoleculeLocalStorage< moleculeStorage > moleculeLocalStorage;
#	endif
__device__ ThreadBlockCellProcessor<
	typeof(moleculeStorage), moleculeStorage,
	typeof(moleculeLocalStorage), moleculeLocalStorage>
		cellProcessor;

#else
__device__ ReferenceCellProcessor<
	typeof(moleculeStorage), moleculeStorage,
	typeof(moleculePairHandler), moleculePairHandler>
		cellProcessor;
#endif

__global__ void processCellPair() {
	const int threadIndex = getThreadIndex();

	const int jobIndex = blockIdx.y * gridDim.x + blockIdx.x;
	if( jobIndex >= DomainTraverser::numJobs ) {
		return;
	}

	int cellIndex = DomainTraverser::getCellIndexFromJobIndex( jobIndex );
	int neighborIndex = DomainTraverser::getNeighborCellIndex( cellIndex );

	CellInfoEx cellA = cellInfoFromCellIndex( cellIndex );
	CellInfoEx cellB = cellInfoFromCellIndex( neighborIndex );

	if( cellA.length == 0 || cellB.length == 0 ) {
		return;
	}

	ThreadBlockCellStats::initThreadLocal( threadIndex );
	cellProcessor.processCellPair( threadIndex, cellA, cellB );

	ThreadBlockCellStats::reduceAndStore( threadIndex, cellIndex, neighborIndex );
}

//__launch_bounds__(BLOCK_SIZE, 2)
__global__ void processCell() {
	const int threadIndex = getThreadIndex();

	int jobIndex = blockIdx.y * gridDim.x + blockIdx.x;
	if( jobIndex >= DomainTraverser::numJobs ) {
		return;
	}

	int cellIndex = DomainTraverser::getInnerCellIndexFromJobIndex(jobIndex);
	CellInfoEx cell = cellInfoFromCellIndex( cellIndex );
	if( cell.length == 0 ) {
		return;
	}

	ThreadBlockCellStats::initThreadLocal( threadIndex );
	cellProcessor.processCell( threadIndex, cell );

	ThreadBlockCellStats::reduceAndStore( threadIndex, cellIndex, cellIndex );
}
#else

__device__ WBCP::CellScheduler *cellScheduler;
__device__ WBCP::CellPairScheduler *cellPairScheduler;

__global__ void createSchedulers() {
	cellScheduler = new WBDP::CellScheduler();
	cellPairScheduler = new WBDP::CellPairScheduler();
}

__global__ void destroySchedulers() {
	delete cellScheduler;
	delete cellPairScheduler;
}

__device__ MoleculeStorage moleculeStorage;
__shared__ ResultLocalStorage< moleculeStorage > resultLocalStorage;
__device__ WBCP::CellProcessor< moleculeStorage, typeof(resultLocalStorage), resultLocalStorage > cellProcessor;

__global__ void processCellPair() {
	const int threadIndex = getThreadIndex();

	__shared__ WBCP::ThreadBlockInfo threadBlockInfo;
	if( threadIndex == 0 ) {
		threadBlockInfo.init();
	}
	__syncthreads();

	do {
		cellPairScheduler->scheduleWarpBlocks( threadBlockInfo );

		while( !threadBlockInfo.warpJobQueue[warpIdx].isEmpty() ) {
			ThreadBlockCellStats::initThreadLocal( threadIndex );

			WBCP::WarpBlockPairInfo warpBlockPairInfo = threadBlockInfo.warpJobQueue[warpIdx].pop();
#ifdef CUDA_HW_CACHE_ONLY
			cellProcessor.processCellPair( warpBlockPairInfo );

			ThreadBlockCellStats::reduceAndStoreWarp( threadIndex, warpBlockPairInfo.warpBlockA.cellIndex );
#else
			cellProcessor.processCellPairWithCache( warpBlockPairInfo );

			ThreadBlockCellStats::reduceAndStoreWarpForPair( threadIndex, warpBlockPairInfo.warpBlockA.cellIndex );
#endif
		}
	} while( threadBlockInfo.hasMoreJobs );

	WARP_PRINTF( "terminating..\n" );
}

__global__ void processCell() {
	// TODO: remove?
	const int threadIndex = getThreadIndex();

	__shared__ WBCP::ThreadBlockInfo threadBlockInfo;
	if( threadIndex == 0 ) {
		threadBlockInfo.init();
	}
	__syncthreads();

	do {
		cellScheduler->scheduleWarpBlocks( threadBlockInfo );

		while( !threadBlockInfo.warpJobQueue[warpIdx].isEmpty() ) {
			ThreadBlockCellStats::initThreadLocal( threadIndex );

			WBCP::WarpBlockPairInfo warpBlockPairInfo = threadBlockInfo.warpJobQueue[warpIdx].pop();
			cellProcessor.processCell( warpBlockPairInfo );

			ThreadBlockCellStats::reduceAndStoreWarp( threadIndex, warpBlockPairInfo.warpBlockA.cellIndex );
		}
	} while( threadBlockInfo.hasMoreJobs );

	WARP_PRINTF( "terminating\n" );
}

#endif

}
