#include <hip/hip_runtime.h>
#include <stdio.h>

class Mutex {
public:
	volatile uint lockValue;

public:
	__device__ void init() {
		lockValue = 0u;
	}

	// locks but supports being interrupted when (signal & bitMask) becomes 0
	// returns true if unlocked must be called
	// returns false if unlocked mustn't be called
	// assumption:
	// the signal can only be reset by a warp/thread that is inside the same semaphore,
	// only the caller can set it
	__device__ bool lock(volatile uint *signal, uint bitMask) {
		while( atomicExch( (uint*) &lockValue, 1u ) != 0u ) {
			if( (*signal & bitMask) == 0 ) {
				return false;
			}
		}

		return true;
	}

	__device__ void unlock() {
		lockValue = 0u;
	}
};

static __device__ Mutex globalMutex;
static volatile __device__ uint globalCounter;

#define WARP_PRINTF( fmt, ... ) printf( "(%i %i) " fmt, blockIdx.x, threadIdx.y, ##__VA_ARGS__ )

static __global__ void init() {
	globalMutex.lockValue = 0;
	globalCounter = 0;
}

static __global__ void printResults() {
	printf( "%i\n", globalCounter );
}

volatile __shared__ uint warpSignals;

static __global__ void kernel() {
	if( threadIdx.x + threadIdx.y == 0 ) {
		printf( "warp 1 lag\n" );
	}

	if( threadIdx.x + threadIdx.y == 0 ) {
		warpSignals = 0u;
	}
	__syncthreads();
	// the __syncthreads and the warp == 1 init is necessary otherwise warpSignals might be reset later on
	// or will not have been initialized properly when it is accessed in other threads

	const uint bitMask = 1 << threadIdx.y;

	if( threadIdx.x == 0 ) {
		atomicOr( (uint*) &warpSignals, bitMask );

		//WARP_PRINTF( "signal: %i\n", warpSignals );
		//WARP_PRINTF( "signal: %i\n", warpSignals );

		if( globalMutex.lock( &warpSignals, bitMask ) ) {
			//WARP_PRINTF( "got lock\n" );

			uint signalSnapshot = warpSignals;

			globalCounter += __popc(signalSnapshot);

			//WARP_PRINTF( "signals\n" );
			atomicXor( (uint*) &warpSignals, signalSnapshot );

			//WARP_PRINTF( "release lock\n" );
			globalMutex.unlock();
		}
		else {
			//WARP_PRINTF( "got signal\n" );
		}
	}
}

void testGlobalSemaphore() {
	init<<<1,1>>>();
	dim3 blockDim = dim3( 32, 16, 1 );
	kernel<<<16, blockDim>>>();
	printResults<<<1,1>>>();
}
