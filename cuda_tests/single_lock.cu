#include <hip/hip_runtime.h>
#include <stdio.h>

class Mutex {
public:
	volatile uint lockValue;

public:
	__device__ void init() {
		lockValue = 0u;
	}

	__device__ void lock() {
		while( atomicExch( (uint*) &lockValue, 1u ) != 0u )
			;
	}

	// locks but supports being interrupted when (signal & bitMask) becomes 0
	// returns true if unlocked must be called
	// returns false if unlocked mustn't be called
	// assumption:
	// the signal can only be reset by a warp/thread that is inside the same semaphore,
	// only the caller can set it
	__device__ bool lock(volatile uint *signal, uint bitMask) {
		while( atomicCAS( (uint*) &lockValue, 0u, 1u ) == 1u ) {
			if( (*signal & bitMask) == 0 ) {
				return false;
			}
		}

		return true;
	}

	__device__ void unlock() {
		lockValue = 0u;
	}
};

static __device__ Mutex globalMutex;
static volatile __device__ uint globalCounter;

#define WARP_PRINTF( fmt, ... ) printf( "(%i %i) " fmt, blockIdx.x, threadIdx.y, ##__VA_ARGS__ )

static __global__ void init() {
	globalMutex.lockValue = 0;
	globalCounter = 0;
}

static __global__ void printResults() {
	printf( "%i\n", globalCounter );
}

static __global__ void kernel() {
	if( threadIdx.x == 0 ) {
		globalMutex.lock();

		//WARP_PRINTF( "got lock\n" );
		globalCounter++;

		//WARP_PRINTF( "incremented globalCounter\n" );
		__threadfence();

		//WARP_PRINTF( "releasing lock\n" );
		globalMutex.unlock();
	}
}

void testSingleLock() {
	init<<<1,1>>>();
	dim3 blockDim = dim3( 32, 16, 1 );
	kernel<<<16, blockDim>>>();
	printResults<<<1,1>>>();
}

