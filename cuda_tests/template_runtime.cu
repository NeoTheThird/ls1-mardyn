/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Template project which demonstrates the basics on how to setup a project 
* example application, doesn't use cutil library.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <iostream>

using namespace std;

bool g_bQATest = false;

#ifdef _WIN32
   #define STRCASECMP  _stricmp
   #define STRNCASECMP _strnicmp
#else
   #define STRCASECMP  strcasecmp
   #define STRNCASECMP strncasecmp
#endif

#define ASSERT(x, msg, retcode) \
    if (!(x)) \
    { \
        cout << msg << " " << __FILE__ << ":" << __LINE__ << endl; \
        return retcode; \
    }

inline hipError_t cutilDeviceSynchronize()
{
	return hipDeviceSynchronize();
}

inline void cutilDeviceReset()
{
	hipDeviceReset();
}

extern void testGlobalSemaphore();
extern void testSingleLock();

int main(int argc, char **argv)
{
    testGlobalSemaphore();
    cutilDeviceReset();

    testSingleLock();
    cutilDeviceReset();
}
